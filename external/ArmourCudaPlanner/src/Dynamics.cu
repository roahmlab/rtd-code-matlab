#include "hip/hip_runtime.h"
#ifndef DYNAMICS_CPP
#define DYNAMICS_CPP

#include "Dynamics.h"

KinematicsDynamics::KinematicsDynamics(BezierCurve* traj_input) {
    traj = traj_input;

    // pre-allocate memory
    links = PZsparseArray(NUM_FACTORS * 3, NUM_TIME_STEPS);
    mass_nominal_arr = PZsparseArray(NUM_JOINTS, 1);
    mass_uncertain_arr = PZsparseArray(NUM_JOINTS, 1);
    I_nominal_arr = PZsparseArray(NUM_JOINTS, 1);
    I_uncertain_arr = PZsparseArray(NUM_JOINTS, 1);
    u_nom = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    u_nom_int = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    f_c_int = PZsparseArray(1,NUM_TIME_STEPS);
    n_c_int = PZsparseArray(1,NUM_TIME_STEPS);
    f_c_nom = PZsparseArray(1,NUM_TIME_STEPS);
    n_c_nom = PZsparseArray(1,NUM_TIME_STEPS);
    r = PZsparseArray(NUM_FACTORS, 1);
    Mr = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);

    // initialize robot properties
    for (int i = 0; i < NUM_JOINTS; i++) {
        trans_matrix(i, 0) = Eigen::MatrixXd::Zero(3, 1);
        trans_matrix(i, 0)(0) = trans[3 * i];
        trans_matrix(i, 0)(1) = trans[3 * i + 1];
        trans_matrix(i, 0)(2) = trans[3 * i + 2];

        com_matrix(i, 0) = Eigen::MatrixXd::Zero(3, 1);
        com_matrix(i, 0)(0) = com[3 * i];
        com_matrix(i, 0)(1) = com[3 * i + 1];
        com_matrix(i, 0)(2) = com[3 * i + 2];

        Eigen::MatrixXd mass_matrix(1, 1);
        mass_matrix(0) = mass[i];
        mass_nominal_arr(i) = PZsparse(mass_matrix);
        mass_uncertain_arr(i) = PZsparse(mass_matrix, mass_uncertainty);

        Eigen::Matrix3d inertia_matrix;
        for (int j = 0; j < 9; j++) {
            inertia_matrix(j) = inertia[i * 9 + j]; // This may not be right...
        }
        I_nominal_arr(i) = PZsparse(inertia_matrix);
        I_uncertain_arr(i) = PZsparse(inertia_matrix, inertia_uncertainty);

        if (i < NUM_FACTORS) {
            r(i) = PZsparse(0, Interval(-eps, eps));
        }
    }

    trans_matrix(NUM_JOINTS, 0) = Eigen::MatrixXd::Zero(3, 1);
    trans_matrix(NUM_JOINTS, 0)(0) = trans[3 * NUM_JOINTS];
    trans_matrix(NUM_JOINTS, 0)(1) = trans[3 * NUM_JOINTS + 1];
    trans_matrix(NUM_JOINTS, 0)(2) = trans[3 * NUM_JOINTS + 2];

    // define original link PZs
    links = PZsparseArray(NUM_JOINTS, NUM_TIME_STEPS);

    for (int i = 0; i < NUM_JOINTS; i++) {
        PZsparseArray link(3, 1);

        for (int j = 0; j < 3; j++) {
            uint64_t degree[1][NUM_FACTORS * 6] = {0};
            degree[0][NUM_FACTORS * (j + 1)] = 1; // use qde, qdae, qdde for x, y, z generator
            double temp = link_zonotope_generators[i][j];
            link(j, 0) = PZsparse(link_zonotope_center[i][j], &temp, degree, 1);
        }

        links(i, 0) = stack(link);

        for (int j = 1; j < NUM_TIME_STEPS; j++) {
            links(i, j) = links(i, 0);
        }
    }
}

void KinematicsDynamics::fk(uint t_ind) {
    PZsparse FK_R = PZsparse(0, 0, 0); // identity matrix
    PZsparse FK_T(3, 1);
    int j = 0;

    for (int i = 0; i < NUM_JOINTS; i++) {
        PZsparse P(trans_matrix(i, 0));
        
        FK_T = FK_T + FK_R * P;
        FK_R = FK_R * traj->R(i, t_ind);
        
        links(i, t_ind) = FK_R * links(i, t_ind) + FK_T;
    }
}

void KinematicsDynamics::rnea(uint t_ind,
                              PZsparseArray& mass_arr,
                              PZsparseArray& I_arr,
                              PZsparseArray& u,
                              PZsparseArray& f_c,
                              PZsparseArray& n_c,
                              bool setGravity) {
    PZsparse w(3, 1);
    PZsparse wdot(3, 1);
    PZsparse w_aux(3, 1);
    PZsparse linear_acc(3, 1);

    PZsparseArray F(NUM_JOINTS, 1);
    PZsparseArray N(NUM_JOINTS, 1);

    if (setGravity) { // set gravity
        // directly modify the center of the PZ instance
        linear_acc.center(2) = gravity;
    }

    // RNEA forward recursion
    for (int i = 0; i < NUM_JOINTS; i++) {
        // NOTE:
        // This is just a simplified implementation!!!
        // We assume all fixed joints are at the end and the revolute joints are consecutive
        if (axes[i] != 0) { // revolute joints
            // line 16
            linear_acc = traj->R_t(i, t_ind) * (linear_acc 
                                                 + cross(wdot, trans_matrix(i, 0)) 
                                                 + cross(w, cross(w_aux, trans_matrix(i, 0))));

            // line 13
            w = traj->R_t(i, t_ind) * w;
            w.addOneDimPZ(traj->qd_des(i, t_ind), abs(axes[i]) - 1, 0);

            // line 14
            w_aux = traj->R_t(i, t_ind) * w_aux;

            // line 15
            wdot = traj->R_t(i, t_ind) * wdot;

            PZsparse temp(3, 1); // temp = joint_vel(robot_params.q_index(i))*z(:,i)
            temp.addOneDimPZ(traj->qd_des(i, t_ind), abs(axes[i]) - 1, 0);

            wdot = wdot + cross(w_aux, temp);

            wdot.addOneDimPZ(traj->qdda_des(i, t_ind), abs(axes[i]) - 1, 0);

            // line 14
            w_aux.addOneDimPZ(traj->qda_des(i, t_ind), abs(axes[i]) - 1, 0);
        }
        else { // fixed joints
            // line 16

            // PZsparse test1 = (linear_acc 
            //                                      + cross(wdot, trans_matrix(i, 0)) 
            //                                      + cross(w, cross(w_aux, trans_matrix(i, 0))));
            // cout << "PZ1" << endl << test1 << endl;
            // PZsparse test2 = traj->R_t(i, t_ind);
            // cout << "PZ2" << endl << test2 << endl;
            
            linear_acc = traj->R_t(i, t_ind) * (linear_acc 
                                                 + cross(wdot, trans_matrix(i, 0)) 
                                                 + cross(w, cross(w_aux, trans_matrix(i, 0))));

            // line 13
            w = traj->R_t(i, t_ind) * w;

            // line 14
            w_aux = traj->R_t(i, t_ind) * w_aux;

            // line 15
            wdot = traj->R_t(i, t_ind) * wdot;
        }

        // line 23 & 27
        F(i, 0) = mass_arr(i, 0) * (linear_acc
                                     + cross(wdot, com_matrix(i, 0))
                                     + cross(w, cross(w_aux, com_matrix(i, 0))));

        // line 29
        N(i, 0) = I_arr(i, 0) * wdot + cross(w_aux, (I_arr(i, 0) * w));
    }

    PZsparse f(3, 1);
    PZsparse n(3, 1);

    // RNEA reverse recursion
    for (int i = NUM_JOINTS - 1; i >= 0; i--) {
        // line 29
        n = N(i, 0)
            + traj->R(i + 1, t_ind) * n
            + cross(com_matrix(i, 0), F(i, 0))
            + cross(trans_matrix(i + 1, 0), traj->R(i + 1, t_ind) * f);

        // line 28
        f = traj->R(i + 1, t_ind) * f + F(i, 0);

        if (axes[i] != 0) {
            u(i, t_ind) = n(abs(axes[i]) - 1, 0);

            u(i, t_ind) = u(i, t_ind) + armature[i] * traj->qdda_des(i, t_ind);

            u(i, t_ind) = u(i, t_ind) + damping[i] * traj->qd_des(i, t_ind);

            // friction is directly cut on the torque limits
        }

        if (i == NUM_JOINTS - 1) {
            f_c(0,t_ind) = f; // not sure how to assign these
            n_c(0,t_ind) = n; // not sure how to assign these
            // note: should change this at some point to be 
            // specifically for a specified list of contact 
            // joints and not just the last joint.
        }
    }
}

#endif