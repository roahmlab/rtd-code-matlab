#include "hip/hip_runtime.h"
#include "NLPclass.h"
#include "BufferPath.h"

const std::string inputext = ".in";
const std::string outputext1 = ".out";
const std::string outputext2 = ".joint_position_center";
const std::string outputext3 = ".joint_position_radius";
const std::string outputext4 = ".control_input_radius";
const std::string outputext5 = ".constraints";

int main(int argc, char **argv) {
/*
Section I:
    Parse input
    There is no check and warning, so be careful!
*/
    // Here is an example of required input
    // double q0[NUM_FACTORS] = {0.6543, -0.0876, -0.4837, -1.2278, -1.5735, -1.0720, 0};
    // double qd0[NUM_FACTORS] = {0, 0, 0, 0, 0, 0, 0};
    // double qdd0[NUM_FACTORS] = {0, 0, 0, 0, 0, 0, 0};
    // double q_des[NUM_FACTORS] = {0.6831, 0.009488, -0.2471, -0.9777, -1.414, -0.9958, 0};

    // const int num_obstacles = 10;
    // const double obstacles[num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {-0.28239,  -0.33281, 0.88069, 0.069825, 0, 0, 0,  0.09508, 0, 0, 0, 0.016624,
    //                                                                             -0.19033,  0.035391,  1.3032,  0.11024, 0, 0, 0, 0.025188, 0, 0, 0, 0.014342,
    //                                                                             0.67593, -0.085841, 0.43572,  0.17408, 0, 0, 0,  0.07951, 0, 0, 0,  0.18012,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             -0.28239,  -0.33281, 0.88069, 0.069825, 0, 0, 0,  0.09508, 0, 0, 0, 0.016624,
    //                                                                             -0.19033,  0.035391,  1.3032,  0.11024, 0, 0, 0, 0.025188, 0, 0, 0, 0.014342,
    //                                                                             0.67593, -0.085841, 0.43572,  0.17408, 0, 0, 0,  0.07951, 0, 0, 0,  0.18012,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981};

    // Parse input
    if(argc != 2) {
        WARNING_PRINT("        CUDA & C++: Missing input argument !\n");
        throw;
    }
    std::string filename = std::string(argv[1]);

    // declare this first and make sure we always have a new output
    std::ofstream outputstream1(pathname + filename + outputext1);

    Eigen::VectorXd q0(NUM_FACTORS); q0.setZero();
    Eigen::VectorXd qd0(NUM_FACTORS); qd0.setZero();
    Eigen::VectorXd qdd0(NUM_FACTORS); qdd0.setZero();
    Eigen::VectorXd q_des(NUM_FACTORS); q_des.setZero();

    int num_obstacles = 0;
    double obstacles[MAX_OBSTACLE_NUM * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {0.0};

    std::ifstream inputstream(pathname + filename + inputext);
    if (!inputstream.is_open()) {
        WARNING_PRINT("        CUDA & C++: Error reading input files !\n");
        outputstream1 << -1;
        outputstream1.close();
        throw;
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> q0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> qd0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> qdd0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> q_des[i];
    }
    inputstream >> num_obstacles;
    if (num_obstacles > MAX_OBSTACLE_NUM || num_obstacles < 0) {
        WARNING_PRINT("Number of obstacles larger than MAX_OBSTACLE_NUM !\n");
        outputstream1 << -1;
        outputstream1.close();
        throw;
    }
    if (num_obstacles > 0) {
        for (int i = 0; i < num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3; i++) {
            inputstream >> obstacles[i];
        }
    }

    inputstream.close();

    double t_plan = 1.0; // optimize the distance between q_des and the desired trajectories at t_plan
     
    /*
Section II:
    Initialize all polynomial zonotopes, including links and torques
*/
    Obstacles O(obstacles, num_obstacles); 

    auto start1 = std::chrono::high_resolution_clock::now();

    omp_set_num_threads(NUM_THREADS);
    int openmp_s_ind = 0; // openmp loop index

    /*
    Section II.A: Create JRS online
    */
    BezierCurve traj(q0, qd0, qdd0);

    try {
        #pragma omp parallel for shared(traj) private(openmp_s_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(openmp_s_ind = 0; openmp_s_ind < NUM_TIME_STEPS; openmp_s_ind++) {
            traj.makePolyZono(openmp_s_ind);
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error creating JRS! Check previous error message!");
        return -1;
    }

    /*
    Section II.B: Compute link PZs and nominal torque PZs
    */
    KinematicsDynamics kd(&traj);
    Eigen::Matrix<double, 3, 3 + 3> link_independent_generators[NUM_TIME_STEPS * NUM_JOINTS];

    try {
        #pragma omp parallel for shared(kd, link_independent_generators) private(openmp_s_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(openmp_s_ind = 0; openmp_s_ind < NUM_TIME_STEPS; openmp_s_ind++) {
            // compute link PZs through forward kinematics
            kd.fk(openmp_s_ind);

            // reduce non-only-k-dependent generators so that slice takes less time
            for (int i = 0; i < NUM_JOINTS; i++) {
                link_independent_generators[openmp_s_ind * NUM_JOINTS + i] = kd.links(i, openmp_s_ind).reduce_link_PZ();
            }

            // compute nominal torque
            kd.rnea_nominal(openmp_s_ind);

            // compute interval torque
            kd.rnea_interval(openmp_s_ind);

            // compute max disturbance (stored in u_nom_int)
            for (int i = 0; i < NUM_FACTORS; i++) {
                kd.u_nom_int(i, openmp_s_ind) = kd.u_nom_int(i, openmp_s_ind) - kd.u_nom(i, openmp_s_ind);
            }

            // reduce non-only-k-dependent generators so that slice takes less time
            for (int i = 0; i < NUM_FACTORS; i++) {
                kd.u_nom(i, openmp_s_ind).reduce();
            }
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error computing link PZs and nominal torque PZs! Check previous error message!");
        return -1;
    }

    /*
    Section II.C: Compute robust input bound
    */
    // the radius of the torque PZs
    Eigen::MatrixXd torque_radius(NUM_FACTORS, NUM_TIME_STEPS);
    torque_radius.setZero();

    try {
        for(int t_ind = 0; t_ind < NUM_TIME_STEPS; t_ind++) {
            // (1) add the bound of robust input (||v||)
            Interval rho_max_temp = Interval(0.0);
            for (int i = 0; i < NUM_FACTORS; i++) {
                // compute norm of disturbance
                MatrixXInt temp = kd.u_nom_int(i, t_ind).toInterval(); // should be a 1-dim Interval
                rho_max_temp += temp(0) * temp(0);

                torque_radius(i, t_ind) = alpha * (M_max - M_min) * eps + 0.5 * max(abs(temp(0).lower()), abs(temp(0).upper()));
            }
            rho_max_temp = sqrt(rho_max_temp);
            
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += 0.5 * rho_max_temp.upper();
            }

            // (2) add the radius of the nominal input PZ (after reducing)
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += kd.u_nom(i, t_ind).independent(0);
            }

            // (3) add friction
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += friction[i];
            }

            // so that torque_radius would be the radius of the total control input PZ from now
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error computing torque PZs! Check previous error message!");
        return -1;
    }

    /*
    Section II.D: Buffer obstacles and initialize collision checking hyperplanes
    */
    try {
        O.initializeHyperPlane(link_independent_generators);
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error initializing collision checking hyperplanes! Check previous error message!");
        return -1;
    }

    auto stop1 = std::chrono::high_resolution_clock::now();
    auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start1);
    cout << "        CUDA & C++: Time taken by generating reachable sets: " << duration1.count() << " milliseconds" << endl;

/*
Section III:
    Solve the optimization problem using IPOPT
*/
    auto start2 = std::chrono::high_resolution_clock::now();

    SmartPtr<armtd_NLP> mynlp = new armtd_NLP();
    try {
	    mynlp->set_parameters(q_des, t_plan, &traj, &kd, &torque_radius, &O);
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error initializing Ipopt! Check previous error message!");
        return -1;
    }

    SmartPtr<IpoptApplication> app = IpoptApplicationFactory();

    app->Options()->SetNumericValue("tol", IPOPT_OPTIMIZATION_TOLERANCE);
	app->Options()->SetNumericValue("max_cpu_time", IPOPT_MAX_CPU_TIME);
	app->Options()->SetIntegerValue("print_level", IPOPT_PRINT_LEVEL);
    app->Options()->SetStringValue("mu_strategy", IPOPT_MU_STRATEGY);
    app->Options()->SetStringValue("linear_solver", IPOPT_LINEAR_SOLVER);
	app->Options()->SetStringValue("hessian_approximation", "limited-memory");

    // For gradient checking
    // app->Options()->SetStringValue("output_file", "ipopt.out");
    // app->Options()->SetStringValue("derivative_test", "first-order");
    // app->Options()->SetNumericValue("derivative_test_perturbation", 1e-8);
    // app->Options()->SetNumericValue("derivative_test_tol", 1e-6);

    // Initialize the IpoptApplication and process the options
    ApplicationReturnStatus status;
    status = app->Initialize();
    if( status != Solve_Succeeded ) {
		WARNING_PRINT("Error during initialization!");
        outputstream1 << -1 << '\n';
        outputstream1.close();
        throw;
    }

    try {
        // Ask Ipopt to solve the problem
        status = app->OptimizeTNLP(mynlp);
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error solving optimization problem! Check previous error message!");
        return -1;
    }
	
    auto stop2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::milliseconds>(stop2 - start2);

    if (status == Maximum_CpuTime_Exceeded) {
        cout << "        CUDA & C++: Ipopt maximum CPU time exceeded!\n";
    }
    
    if (status == Invalid_Option) {
        cout << "        CUDA & C++: Cannot find HSL library! Need to put libcoinhsl.so in proper path!\n";
    }
    else {
        cout << "        CUDA & C++: Time taken by Ipopt: " << duration2.count() << " milliseconds" << endl;
    }

/*
Section IV:
    Prepare output
*/
    // set precision to 10 decimal digits
    outputstream1 << std::setprecision(10);

    // output k_opt
    if (mynlp->feasible) {
        for (int i = 0; i < NUM_FACTORS; i++) {
            outputstream1 << mynlp->solution[i] << '\n';
        }
    }
    else {
        outputstream1 << -1 << '\n';
    }

    // output time cost (in milliseconds) in C++
    outputstream1 << duration1.count() + duration2.count();
    outputstream1.close();

    // output FRS and other information, you can comment them if they are unnecessary
    std::ofstream outputstream2(pathname + filename + outputext2);
    outputstream2 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_JOINTS; j++) {
            for (int l = 0; l < 3; l++) {
                outputstream2 << mynlp->link_sliced_center[i * NUM_JOINTS + j](l) << ' ';
            }
            outputstream2 << '\n';
        }
    }
    outputstream2.close();

    std::ofstream outputstream3(pathname + filename + outputext3);
    outputstream3 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_JOINTS; j++) {
            for (int k = 0; k < 3; k++) {
                for (int l = 0; l < 3 + 3; l++) {
                    outputstream3 << link_independent_generators[i * NUM_JOINTS + j](k, l) << ' ';
                }
                outputstream3 << '\n';
            }
        }
    }
    outputstream3.close();

    std::ofstream outputstream4(pathname + filename + outputext4);
    outputstream4 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_FACTORS; j++) {
            outputstream4 << torque_radius(j, i) << ' '; // this is radius of final control input
        }
        outputstream4 << '\n';
    }
    outputstream4.close();

    std::ofstream outputstream5(pathname + filename + outputext5);
    outputstream5 << std::setprecision(6);
    for (int i = 0; i < mynlp->constraint_number; i++) {
        outputstream5 << mynlp->g_copy[i] << '\n';
    }
    outputstream5.close();

    return 0;
}
