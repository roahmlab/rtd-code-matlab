#include "hip/hip_runtime.h"
#include "NLPclass.h"
#include "BufferPath.h"

const std::string inputext = ".in";
const std::string outputext1 = ".out";
const std::string outputext2 = "_joint_position_center.out";
const std::string outputext3 = "_joint_position_radius.out";
const std::string outputext4 = "_control_input_radius.out";
const std::string outputext5 = "_constraints.out";
const std::string outputext6 = "_wrench_values.out";
const std::string outputext7 = "_force_constraint_radius.out";

int main(int argc, char **argv) {
/*
Section I:
    Parse input
    There is no check and warning, so be careful!
*/
    // Here is an example of required input
    // double q0[NUM_FACTORS] = {0.6543, -0.0876, -0.4837, -1.2278, -1.5735, -1.0720, 0};
    // double qd0[NUM_FACTORS] = {0, 0, 0, 0, 0, 0, 0};
    // double qdd0[NUM_FACTORS] = {0, 0, 0, 0, 0, 0, 0};
    // double q_des[NUM_FACTORS] = {0.6831, 0.009488, -0.2471, -0.9777, -1.414, -0.9958, 0};

    // const int num_obstacles = 10;
    // const double obstacles[num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {-0.28239,  -0.33281, 0.88069, 0.069825, 0, 0, 0,  0.09508, 0, 0, 0, 0.016624,
    //                                                                             -0.19033,  0.035391,  1.3032,  0.11024, 0, 0, 0, 0.025188, 0, 0, 0, 0.014342,
    //                                                                             0.67593, -0.085841, 0.43572,  0.17408, 0, 0, 0,  0.07951, 0, 0, 0,  0.18012,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             -0.28239,  -0.33281, 0.88069, 0.069825, 0, 0, 0,  0.09508, 0, 0, 0, 0.016624,
    //                                                                             -0.19033,  0.035391,  1.3032,  0.11024, 0, 0, 0, 0.025188, 0, 0, 0, 0.014342,
    //                                                                             0.67593, -0.085841, 0.43572,  0.17408, 0, 0, 0,  0.07951, 0, 0, 0,  0.18012,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981};
    // Parse input
    if(argc != 2) {
        WARNING_PRINT("        CUDA & C++: Missing input argument !\n");
        throw;
    }
    std::string filename = std::string(argv[1]);
    
    // declare this first and make sure we always have a new output
    std::ofstream outputstream1(pathname + filename + outputext1);

    Eigen::VectorXd q0(NUM_FACTORS); q0.setZero();
    Eigen::VectorXd qd0(NUM_FACTORS); qd0.setZero();
    Eigen::VectorXd qdd0(NUM_FACTORS); qdd0.setZero();
    Eigen::VectorXd q_des(NUM_FACTORS); q_des.setZero();

    int num_obstacles = 0;
    double obstacles[MAX_OBSTACLE_NUM * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {0.0};

    std::ifstream inputstream(pathname + filename + inputext);
    if (!inputstream.is_open()) {
        WARNING_PRINT("        CUDA & C++: Error reading input files !\n");
        outputstream1 << -1;
        outputstream1.close();
        throw;
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> q0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> qd0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> qdd0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> q_des[i];
    }
    inputstream >> num_obstacles;
    if (num_obstacles > MAX_OBSTACLE_NUM || num_obstacles < 0) {
        WARNING_PRINT("Number of obstacles larger than MAX_OBSTACLE_NUM !\n");
        outputstream1 << -1;
        outputstream1.close();
        throw;
    }
    if (num_obstacles > 0) {
        for (int i = 0; i < num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3; i++) {
            inputstream >> obstacles[i];
        }
    }

    inputstream.close();

    double t_plan = DURATION; // optimize the distance between q_des and the desired trajectories at t_plan
    // Kinova Hardware Demo Values: u_s = 0.609382421; surf_rad =  0.058/2;
    double u_s = 0.609382421; // 0.5; // static coefficient of friction between tray and object
    double surf_rad =  0.058 / 2; // 0.0762; // RADIUS of contact area between tray and object (area assumed to be circular) 
    // Note: might want to change this to be input to the C++ code from matlab?
    
    /*
Section II:
    Initialize all polynomial zonotopes, including links and torques
*/
    Obstacles O;
    O.initialize(obstacles, num_obstacles); 

    auto start1 = std::chrono::high_resolution_clock::now();

    omp_set_num_threads(NUM_THREADS);
    int openmp_t_ind = 0; // openmp loop index

    /*
    Section II.A: Create JRS online
    */
    BezierCurve traj(q0, qd0, qdd0);

    try {
        #pragma omp parallel for shared(traj) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
            traj.makePolyZono(openmp_t_ind);
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error creating JRS! Check previous error message!");
        return -1;
    }

    /*
    Section II.B: Compute link PZs and nominal torque PZs
    */
    KinematicsDynamics kd(&traj);
    Eigen::Matrix<double, 3, 3 + 3> link_independent_generators[NUM_TIME_STEPS * NUM_JOINTS];

    try {
        #pragma omp parallel for shared(kd, link_independent_generators) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
            // compute link PZs through forward kinematics
            kd.fk(openmp_t_ind);

            // reduce non-only-k-dependent generators so that slice takes less time
            for (int i = 0; i < NUM_JOINTS; i++) {
                link_independent_generators[openmp_t_ind * NUM_JOINTS + i] = kd.links(i, openmp_t_ind).reduce_link_PZ();
            }

            // compute nominal torque
            kd.rnea_nominal(openmp_t_ind);

            // compute interval torque
            kd.rnea_interval(openmp_t_ind);

            // compute max disturbance (stored in u_nom_int)
            for (int i = 0; i < NUM_FACTORS; i++) {
                kd.u_nom_int(i, openmp_t_ind) = kd.u_nom_int(i, openmp_t_ind) - kd.u_nom(i, openmp_t_ind);
            }

            // reduce non-only-k-dependent generators so that slice takes less time
            for (int i = 0; i < NUM_FACTORS; i++) {
                kd.u_nom(i, openmp_t_ind).reduce();
            }
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error computing link PZs and nominal torque PZs! Check previous error message!");
        return -1;
    }


    // kd.f_c(127)(2,0).slice(storage*, x) takes the last 3x1 and then takes the third element and slices it
    // in NLPclass use -> instead of . 
    
    
    /*
    Section II.C: Compute robust input bound
    */
    // the radius of the torque PZs
    Eigen::MatrixXd torque_radius(NUM_FACTORS, NUM_TIME_STEPS);
    torque_radius.setZero();

    try {
        for(int t_ind = 0; t_ind < NUM_TIME_STEPS; t_ind++) {
            // (1) add the bound of robust input (||v||)
            Interval rho_max_temp = Interval(0.0);
            for (int i = 0; i < NUM_FACTORS; i++) {
                // compute norm of disturbance
                MatrixXInt temp = kd.u_nom_int(i, t_ind).toInterval(); // should be a 1-dim Interval
                rho_max_temp += temp(0) * temp(0);

                torque_radius(i, t_ind) = alpha * (M_max - M_min) * eps + 0.5 * max(abs(temp(0).lower()), abs(temp(0).upper()));
            }
            rho_max_temp = sqrt(rho_max_temp);
            
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += 0.5 * rho_max_temp.upper();
            }

            // (2) add the radius of the nominal input PZ (after reducing)
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += kd.u_nom(i, t_ind).independent(0);
            }

            // (3) add friction
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += friction[i];
            }

            // so that torque_radius would be the radius of the total control input PZ from now
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error computing torque PZs! Check previous error message!");
        return -1;
    }

    /*
    Section II.D: Buffer obstacles and initialize collision checking hyperplanes
    */
    try {
        O.initializeHyperPlane(link_independent_generators);
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error initializing collision checking hyperplanes! Check previous error message!");
        return -1;
    }

    auto stop1 = std::chrono::high_resolution_clock::now();
    auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start1);
    cout << "        CUDA & C++: Time taken by generating reachable sets: " << duration1.count() << " milliseconds" << endl;

/*
Section III:
    Solve the optimization problem using IPOPT
*/
    auto start2 = std::chrono::high_resolution_clock::now();

    SmartPtr<armtd_NLP> mynlp = new armtd_NLP();
    try {
	    mynlp->set_parameters(q_des, t_plan, &traj, &kd, &torque_radius, &O, u_s, surf_rad);
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error initializing Ipopt! Check previous error message!");
        return -1;
    }

    SmartPtr<IpoptApplication> app = IpoptApplicationFactory();

    app->Options()->SetNumericValue("tol", IPOPT_OPTIMIZATION_TOLERANCE);
	app->Options()->SetNumericValue("max_cpu_time", IPOPT_MAX_CPU_TIME);
	app->Options()->SetIntegerValue("print_level", IPOPT_PRINT_LEVEL);
    app->Options()->SetStringValue("mu_strategy", IPOPT_MU_STRATEGY);
    app->Options()->SetStringValue("linear_solver", IPOPT_LINEAR_SOLVER);
	app->Options()->SetStringValue("hessian_approximation", "limited-memory");

    // For gradient checking
    // app->Options()->SetStringValue("output_file", "ipopt.out");
    // app->Options()->SetStringValue("derivative_test", "first-order");
    // app->Options()->SetNumericValue("derivative_test_perturbation", 1e-8);
    // app->Options()->SetNumericValue("derivative_test_tol", 1e-6);

    // Initialize the IpoptApplication and process the options
    ApplicationReturnStatus status;
    status = app->Initialize();
    if( status != Solve_Succeeded ) {
		WARNING_PRINT("Error during initialization!");
        outputstream1 << -1 << '\n';
        outputstream1.close();
        throw;
    }

    try {
        // Ask Ipopt to solve the problem
        status = app->OptimizeTNLP(mynlp);
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error solving optimization problem! Check previous error message!");
        return -1;
    }
	
    auto stop2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::milliseconds>(stop2 - start2);

    if (status == Maximum_CpuTime_Exceeded) {
        cout << "        CUDA & C++: Ipopt maximum CPU time exceeded!\n";
    }
    
    if (status == Invalid_Option) {
        cout << "        CUDA & C++: Cannot find HSL library! Need to put libcoinhsl.so in proper path!\n";
    }
    else {
        cout << "        CUDA & C++: Time taken by Ipopt: " << duration2.count() << " milliseconds" << endl;
    }

/*
Section IV:
    Prepare output
*/
    // set precision to 10 decimal digits
    outputstream1 << std::setprecision(10);

    // output k_opt
    if (mynlp->feasible) {
        for (int i = 0; i < NUM_FACTORS; i++) {
            outputstream1 << mynlp->solution[i] << '\n';
        }
    }
    else {
        outputstream1 << -1 << '\n';
    }

    // output time cost (in milliseconds) in C++
    outputstream1 << duration1.count() + duration2.count();
    outputstream1.close();

    // output FRS and other information, you can comment them if they are unnecessary
    std::ofstream outputstream2(pathname + filename + outputext2);
    outputstream2 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_JOINTS; j++) {
            for (int l = 0; l < 3; l++) {
                outputstream2 << mynlp->link_sliced_center[i * NUM_JOINTS + j](l) << ' ';
            }
            outputstream2 << '\n';
        }
        outputstream2 << '\n';
    }
    outputstream2.close();

    std::ofstream outputstream3(pathname + filename + outputext3);
    outputstream3 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_JOINTS; j++) {
            for (int k = 0; k < 3; k++) {
                for (int l = 0; l < 3 + 3; l++) {
                    outputstream3 << link_independent_generators[i * NUM_JOINTS + j](k, l) << ' ';
                }
                outputstream3 << '\n';
            }
            outputstream3 << '\n';
        }
        outputstream3 << '\n';
    }
    outputstream3.close();

    std::ofstream outputstream4(pathname + filename + outputext4);
    outputstream4 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_FACTORS; j++) {
            outputstream4 << torque_radius(j, i) << ' '; // this is radius of final control input
        }
        outputstream4 << '\n';
    }
    outputstream4.close();

    std::ofstream outputstream5(pathname + filename + outputext5);
    outputstream5 << std::setprecision(6);
    for (int i = 0; i < mynlp->constraint_number; i++) {
        outputstream5 << mynlp->g_copy[i] << '\n';
    }
    outputstream5.close();

    // need to change variables that are being output
    // need to add calculation of lower bound to NLPclass.cu and a way to store it
    // outputting wrench center values and radii
    std::ofstream outputstream6(pathname + filename + outputext6);
    outputstream6 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < 3; j++) {
            outputstream6 << mynlp -> force_value_center(j, i) << ' ';
        }
        for (int j = 0; j < 3; j++) {
            outputstream6 << mynlp -> moment_value_center(j, i) << ' ';
        }
        for (int j = 0; j < 3; j++) {
            outputstream6 << mynlp -> force_value_radii(j, i) << ' ';
        }
        for (int j = 0; j < 3; j++) {
            outputstream6 << mynlp -> moment_value_radii(j, i) << ' ';
        }
        outputstream6 << '\n';
    }
    outputstream6.close();

    // outputting contact constraint values
    std::ofstream outputstream7(pathname + filename + outputext7);
    outputstream7 << std::setprecision(10);
    for (int i = 0; i < 3*NUM_TIME_STEPS; i++) {
        outputstream7 << mynlp -> force_constraint_ub[i] << ' ' << mynlp -> force_constraint_lb[i] << ' ';
        outputstream7 << '\n';
    }
    outputstream7.close();

    return 0;
}