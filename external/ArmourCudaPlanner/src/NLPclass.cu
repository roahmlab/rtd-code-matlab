#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU

#include "NLPclass.h"

// constructor
armtd_NLP::armtd_NLP()
{
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    Eigen::VectorXd& q_des_input,
    double t_plan_input,
    const BezierCurve* desired_trajectory_input,
    KinematicsDynamics* kinematics_dynamics_result_input,
    const Eigen::MatrixXd* torque_radius_input,
    Obstacles* obstacles_input,
    double u_s_input,
    double surf_rad_input
 ) 
 {
    q_des = q_des_input;
    t_plan = t_plan_input;
    desired_trajectory = desired_trajectory_input;
    kinematics_dynamics_result = kinematics_dynamics_result_input;
    torque_radius = torque_radius_input;
    obstacles = obstacles_input;
    u_s = u_s_input;
    surf_rad = surf_rad_input;

    constraint_number = NUM_FACTORS * NUM_TIME_STEPS +
                        3*NUM_TIME_STEPS +
                        NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles + 
                        NUM_FACTORS * 4;

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            g_l[i * NUM_FACTORS + j] = -torque_limits[j] + (*torque_radius)(j, i);
            g_u[i * NUM_FACTORS + j] = torque_limits[j] - (*torque_radius)(j, i);
        }
    }    
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;

    //     separation constraint
    // upper bound should be zero and lower bound should be -inf
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     slipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint (not the normal friction law?)
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     tipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    // collision avoidance constraints
    for( Index i = offset; i < offset + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        x[i] = 0.0;

        // try to avoid local minimum
        // x[i] = min(max((q_des[i] - desired_trajectory->q0[i]) / k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

void armtd_NLP::compute(
    bool new_x,
    const Number* x
){

    // check if a new x is passed in
    if (new_x){
        // timing
        // auto start_compute = std::chrono::high_resolution_clock::now();

        // update values

        // compute the constraint values

        // Contact Force Constraints
        Index i;
        #pragma omp parallel for shared(kinematics_dynamics_result, x, link_sliced_center) private(i) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(i = 0; i<NUM_TIME_STEPS; i++){

            for (int m = 0; m < 3; m++) {
                MatrixXInt res1 = kinematics_dynamics_result -> f_c_int(i)(m,0).slice(x);
                force_value_center(m,i) = getCenter(res1(0));
                force_value_radii(m,i) = getRadius(res1(0));
                MatrixXInt res2 = kinematics_dynamics_result -> n_c_int(i)(m,0).slice(x);
                moment_value_center(m,i) = getCenter(res2(0));
                moment_value_radii(m,i) = getRadius(res2(0));
            }

            // Extract the force PZs, slice, and get the centers and radii
            MatrixXInt f_c_x = kinematics_dynamics_result -> f_c_int(i)(0,0).slice(x);
            Number f_c_x_center = getCenter(f_c_x(0));
            Number f_c_x_radius = getRadius(f_c_x(0));

            MatrixXInt f_c_y = kinematics_dynamics_result -> f_c_int(i)(1,0).slice(x);
            Number f_c_y_center = getCenter(f_c_y(0));
            Number f_c_y_radius = getRadius(f_c_y(0));

            MatrixXInt f_c_z = kinematics_dynamics_result -> f_c_int(i)(2,0).slice(x);
            Number f_c_z_center = getCenter(f_c_z(0));
            Number f_c_z_radius = getRadius(f_c_z(0));

            // Extract the moment PZs
            MatrixXInt n_c_x = kinematics_dynamics_result -> n_c_int(i)(0,0).slice(x);
            Number n_c_x_center = getCenter(n_c_x(0));
            Number n_c_x_radius = getRadius(n_c_x(0));

            MatrixXInt n_c_y = kinematics_dynamics_result -> n_c_int(i)(1,0).slice(x);
            Number n_c_y_center = getCenter(n_c_y(0));
            Number n_c_y_radius = getRadius(n_c_y(0));

            MatrixXInt n_c_z = kinematics_dynamics_result -> n_c_int(i)(2,0).slice(x);
            Number n_c_z_center = getCenter(n_c_z(0));
            Number n_c_z_radius = getRadius(n_c_z(0));

            // compute the numerator of the ZMP point equation
            Eigen::MatrixXd norm_vec(3,1);
            norm_vec << 0,0,1;
            PZsparse ZMP_top = cross(norm_vec,kinematics_dynamics_result->n_c_int(i));
            // extract the x, y and z components, slice by the parameters, then get the centers and radii of independent generators
            // x-component
            MatrixXInt ZMP_top_x = ZMP_top(0,0).slice(x); // ->?
            Number ZMP_top_x_center = getCenter(ZMP_top_x(0));
            Number ZMP_top_x_radius = getRadius(ZMP_top_x(0));
            // y-component
            MatrixXInt ZMP_top_y = ZMP_top(1,0).slice(x); // ->?
            Number ZMP_top_y_center = getCenter(ZMP_top_y(0));
            Number ZMP_top_y_radius = getRadius(ZMP_top_y(0));
            // z-component (for verification, this should be zero always.)
            MatrixXInt ZMP_top_z = ZMP_top(2,0).slice(x); // ->?
            Number ZMP_top_z_center = getCenter(ZMP_top_z(0));
            Number ZMP_top_z_radius = getRadius(ZMP_top_z(0));

            // compute the denominator of the ZMP point equation
            MatrixXInt ZMP_bottom = kinematics_dynamics_result->f_c_int(i)(2,0).slice(x);
            Number ZMP_bottom_center = getCenter(ZMP_bottom(0));
            Number ZMP_bottom_radius = getRadius(ZMP_bottom(0));


            // constraints

            // Separation constraint
            force_constraint_ub[i] = -1*f_c_z_center + f_c_z_radius;
            force_constraint_lb[i] = -1*f_c_z_center - f_c_z_radius;

            Index idx_offset2 = NUM_TIME_STEPS;

            // slipping constraint
            if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ){
                // Note: double check that the center/radius is a number that can be squared
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 2: y negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 3: z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 4: y and z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 5: x negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 6: x and y negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 7: x and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }
            // condition 8: x and y and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(f_c_x_center,2) - 2*f_c_x_radius*f_c_x_center + pow(f_c_x_radius,2) + pow(f_c_y_center,2) - 2*f_c_y_radius*f_c_y_center + pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) + 2*f_c_z_radius*f_c_z_center - pow(f_c_z_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(f_c_x_center,2) + 2*f_c_x_radius*f_c_x_center - pow(f_c_x_radius,2) + pow(f_c_y_center,2) + 2*f_c_y_radius*f_c_y_center - pow(f_c_y_radius,2) - pow(u_s,2) * ( pow(f_c_z_center,2) - 2*f_c_z_radius*f_c_z_center + pow(f_c_z_radius,2)); // checked signs

            }

            idx_offset2 += NUM_TIME_STEPS;
            // tipping constraint

            // condition 1: all positive
            if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center >= 0) ){
                // Note: double check that the center/radius is a number that can be squared
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 2: y negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center >= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 3: z negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 4: y and z negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 5: x negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center >= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 6: x and y negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center >= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 7: x and z negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }
            // condition 8: x and y and z negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center <= 0) ) {
                force_constraint_ub[i+idx_offset2] = pow(ZMP_top_x_center,2) - 2*ZMP_top_x_radius*ZMP_top_x_center + pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) - 2*ZMP_top_y_radius*ZMP_top_y_center + pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) + 2*ZMP_bottom_radius*ZMP_bottom_center - pow(ZMP_bottom_radius,2)); // checked signs
                force_constraint_lb[i+idx_offset2] = pow(ZMP_top_x_center,2) + 2*ZMP_top_x_radius*ZMP_top_x_center - pow(ZMP_top_x_radius,2) + pow(ZMP_top_y_center,2) + 2*ZMP_top_y_radius*ZMP_top_y_center - pow(ZMP_top_y_radius,2) - pow(surf_rad,2) * ( pow(ZMP_bottom_center,2) - 2*ZMP_bottom_radius*ZMP_bottom_center + pow(ZMP_bottom_radius,2)); // checked signs

            }


            // calculate the gradient of the constraints

            // to do: check the index offsets and clean up code and add public variable to store the gradient values
            // to do: remove duplicate code from other functions and properly access and assign these variables in those functions

            // Contact Force Constraints

            // gradients
            // storage for the gradients
            Number f_c_x_grad[NUM_FACTORS];
            Number f_c_y_grad[NUM_FACTORS];
            Number f_c_z_grad[NUM_FACTORS];
            Number ZMP_top_x_grad[NUM_FACTORS];
            Number ZMP_top_y_grad[NUM_FACTORS];
            Number ZMP_bottom_grad[NUM_FACTORS];
            // calculate the gradients
            kinematics_dynamics_result->f_c_int(i)(0,0).slice(f_c_x_grad, x);
            kinematics_dynamics_result->f_c_int(i)(1,0).slice(f_c_y_grad, x);
            kinematics_dynamics_result->f_c_int(i)(2,0).slice(f_c_z_grad, x);
            ZMP_top(0,0).slice(ZMP_top_x_grad, x);
            ZMP_top(1,0).slice(ZMP_top_y_grad, x);
            kinematics_dynamics_result->f_c_int(i)(2,0).slice(ZMP_bottom_grad, x); // same as f_c_z_grad?

            // Separation constraint gradient
            for (int j = 0;j<NUM_FACTORS;j++) {
                force_constraint_gradient[i*NUM_FACTORS+j] = -1*f_c_z_grad[j];
            }

            Index grad_idx_offset = NUM_TIME_STEPS*NUM_FACTORS;
            // Slipping Constraint
            // calculate constraint gradient, depends on the signs of the centers like constraint itself does.
            if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ){
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] + 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] + 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] - 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 2: y negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] + 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] - 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] - 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 3: z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] + 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] + 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] + 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 4: y and z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] + 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] - 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] + 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 5: x negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] - 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] + 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] - 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 6: x and y negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] - 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] - 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] - 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 7: x and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] - 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] + 2*f_c_y_radius*f_c_y_grad[j] - pow(u_s,2) * ( 2*f_c_z_center*f_c_z_grad[j] + 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }
            // condition 8: x and y and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*f_c_x_center*f_c_x_grad[j] - 2*f_c_x_radius*f_c_x_grad[j] + 2*f_c_y_center*f_c_y_grad[j] - 2*f_c_y_radius*f_c_y_grad[j] - u_s*u_s * ( 2*f_c_z_center*f_c_z_grad[j] + 2*f_c_z_radius*f_c_z_grad[j] );
                }
            }

            //    tipping constraint

            grad_idx_offset += NUM_TIME_STEPS*NUM_FACTORS;
            // calculate constraint gradient
            if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center >= 0) ){
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] + 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] + 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] - 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 2: y negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center >= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] + 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] - 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] - 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 3: z negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] + 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] + 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] + 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 4: y and z negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] + 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] - 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] + 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 5: x negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center >= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] - 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] + 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] - 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 6: x and y negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center >= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] - 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] - 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] - 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 7: x and z negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] - 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] + 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] + 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }
            // condition 8: x and y and z negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center <= 0) ) {
                for (int j=0;j<NUM_FACTORS;j++) {
                    force_constraint_gradient[i*NUM_FACTORS+grad_idx_offset+j] = 2*ZMP_top_x_center*ZMP_top_x_grad[j] - 2*ZMP_top_x_radius*ZMP_top_x_grad[j] + 2*ZMP_top_y_center*ZMP_top_y_grad[j] - 2*ZMP_top_y_radius*ZMP_top_y_grad[j] - pow(surf_rad,2) * ( 2*ZMP_bottom_center*ZMP_bottom_grad[j] + 2*ZMP_bottom_radius*ZMP_bottom_grad[j]);
                }
            }

            
        }
        // auto stop_compute = std::chrono::high_resolution_clock::now();
        // auto duration_compute = std::chrono::duration_cast<std::chrono::milliseconds>(stop_compute - start_compute);
        // cout << "        Time Taken to Calculate Compute Function: " << duration_compute.count() << " milliseconds" << endl;

    }
    else{
        // do not update values
    }

}

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // call compute function to update values if necessary
    // cout << "Computing new_x from f function" << endl;
    // auto start_new_x = std::chrono::high_resolution_clock::now();

    compute(new_x,x);

    // auto stop_new_x = std::chrono::high_resolution_clock::now();
    // auto duration_new_x = std::chrono::duration_cast<std::chrono::milliseconds>(stop_new_x - start_new_x);
    // cout << "        Time Taken to Calculate new_x from f grad: " << duration_new_x.count() << " milliseconds" << endl;

    // auto start_nom = std::chrono::high_resolution_clock::now();

    // obj_value = sum((q_plan - q_des).^2);
    obj_value = 0; 
    for(Index i = 0; i < n; i++){
        double q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan); // Bohao question: why pass in t_plan here instead of duration?
        obj_value += pow(q_des[i] - q_plan, 2);
    }

    obj_value *= COST_FUNCTION_OPTIMALITY_SCALE; // needs to change in the gradient as well

    // auto stop_nom = std::chrono::high_resolution_clock::now();
    // auto duration_nom = std::chrono::duration<long, std::nano>(stop_nom - start_nom);
    // cout << "        Time for Evaluating Cost Nominal Term: " << duration_nom.count() << " nanoseconds" << endl;

    // auto start_f = std::chrono::high_resolution_clock::now();

    // // new cost term
    // // loop to pull out slip constraint values
    // for(Index i=0; i<NUM_TIME_STEPS;i++){
    //     // offset by NUM_TIME_STEPS to move past the separation constraint
    //     obj_value += force_constraint_ub[i+NUM_TIME_STEPS];
    // }

    // auto stop_f = std::chrono::high_resolution_clock::now();
    // auto duration_f = std::chrono::duration<long, std::nano>(stop_f - start_f);
    // cout << "        Time for Evaluating Cost Slip Term: " << duration_f.count() << " nanoseconds" << endl;

    // for(Index i = 0; i < NUM_TIME_STEPS; i++){
    //     obj_value += cost_slip_ub[i];
    // }

    

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    // call compute to update if new_x
    // cout << "Computing new_x from f grad function" << endl;
    // auto start_new_x = std::chrono::high_resolution_clock::now();

    compute(new_x,x);

    // auto stop_new_x = std::chrono::high_resolution_clock::now();
    // auto duration_new_x = std::chrono::duration_cast<std::chrono::milliseconds>(stop_new_x - start_new_x);
    // cout << "        Time Taken to Calculate new_x from f grad: " << duration_new_x.count() << " milliseconds" << endl;


    for(Index i = 0; i < n; i++){

        // values is 7x1
        // sum for each time step resulting in 7x1
        // add each element to corresponding element of grad_f?

        double q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan); // Bohao question: why pass in t_plan here instead of duration?
        double dk_q_plan = pow(t_plan,3) * (6 * pow(t_plan,2) - 15 * t_plan + 10);
        grad_f[i] = (2 * (q_plan - q_des[i]) * dk_q_plan * k_range[i]) * COST_FUNCTION_OPTIMALITY_SCALE;
    }

    // auto start_grad = std::chrono::high_resolution_clock::now();

    // new cost term gradient
    Index offset = NUM_TIME_STEPS*NUM_FACTORS; // offset to pass over separation constraint
    for(Index i=0; i<NUM_TIME_STEPS;i++){
        // sum cost_grad_slip and grad_f column-wise
        for(Index j = 0; j < n; j++){
            // grad_f[j] += cost_grad_slip[i];
            grad_f[j] += force_constraint_gradient[i*NUM_FACTORS+offset+j];
        }
    }

    // auto stop_grad = std::chrono::high_resolution_clock::now();
    // auto duration_grad = std::chrono::duration<long, std::nano>(stop_grad - start_grad);
    // cout << "        Time for Evaluating Cost Gradient Slip Term: " << duration_grad.count() << " nanoseconds" << endl;

    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    // cout << "Computing new_x from g function" << endl;
    compute(new_x,x);

    // auto start_g = std::chrono::high_resolution_clock::now();

    Index i;
    #pragma omp parallel for shared(kinematics_dynamics_result, x, g, link_sliced_center) private(i) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
    for(i = 0; i < NUM_TIME_STEPS; i++) {
        for (int k = 0; k < NUM_FACTORS; k++) {
            MatrixXInt res = kinematics_dynamics_result->u_nom(k, i).slice(x);
            g[i * NUM_FACTORS + k] = getCenter(res(0));
        }

        for (int l = 0; l < NUM_JOINTS; l++) {
            MatrixXInt res = kinematics_dynamics_result->links(l, i).slice(x);
            link_sliced_center[i * NUM_JOINTS + l] = getCenter(res);
        }
    }

    // Part 2. force constraints
    // offset by the number of input constraints NUM_TIME_STEPS*NUM_FACTORS
    for(Index i=0; i<3*NUM_TIME_STEPS;i++){
        g[i+NUM_TIME_STEPS*NUM_FACTORS] = force_constraint_ub[i];
    }

    // // For loop to iterate
    // for(int j=0; j<NUM_FACTORS;j++){
    //     cout << x[j] << ", ";
    // }
    // cout << "\n";
    // int out_index = NUM_FACTORS*NUM_TIME_STEPS + NUM_TIME_STEPS;
    // for(int i=0; i<NUM_TIME_STEPS;i++){
    //     cout << g[i+out_index] << "\n";
    // }

    // Part 3. check collision between joint position reachable set and obstacles (in gpu)
    obstacles->linkFRSConstraints(link_sliced_center, nullptr, g + NUM_FACTORS*NUM_TIME_STEPS + 3*NUM_TIME_STEPS, nullptr);

    // Part 4. (position & velocity) state limit constraints
    desired_trajectory->returnJointPositionExtremum(g + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS*NUM_TIME_STEPS + 3*NUM_TIME_STEPS, x);
    desired_trajectory->returnJointVelocityExtremum(g + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2 + NUM_FACTORS*NUM_TIME_STEPS + 3*NUM_TIME_STEPS, x);

    // auto stop_g = std::chrono::high_resolution_clock::now();
    // auto duration_g = std::chrono::duration_cast<std::chrono::milliseconds>(stop_g - start_g);
    // cout << "        Time Taken to Calculate Eval g Function: " << duration_g.count() << " milliseconds" << endl;

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    // cout << "Computing new_x from g grad function" << endl;
    compute(new_x,x); // could this move into the else of the if statement below?

    // auto start_g_grad = std::chrono::high_resolution_clock::now();

    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;
        #pragma omp parallel for shared(kinematics_dynamics_result, x, values, link_sliced_center, dk_link_sliced_center) private(i) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(i = 0; i < NUM_TIME_STEPS; i++) {
            for (int k = 0; k < NUM_FACTORS; k++) {
                kinematics_dynamics_result->u_nom(k, i).slice(values + (i * NUM_FACTORS + k) * NUM_FACTORS, x);
            }

            for (int l = 0; l < NUM_JOINTS; l++) {
                link_sliced_center[i * NUM_JOINTS + l] = getCenter(kinematics_dynamics_result->links(l, i).slice(x));
                kinematics_dynamics_result->links(l, i).slice(dk_link_sliced_center + (i * NUM_JOINTS + l) * NUM_FACTORS, x);
            }
        }

        
        // Part 2. force constraints
        // offset by number of input constraint gradient terms: NUM_TIME_STEPS*NUM_FACTORS*NUM_FACTORS
        for(Index i=0; i<3*NUM_TIME_STEPS*NUM_FACTORS;i++){
            values[i+NUM_TIME_STEPS*NUM_FACTORS*NUM_FACTORS] = force_constraint_gradient[i];
        }

        // Part 3. check collision between joint position reachable set and obstacles (in gpu)
        obstacles->linkFRSConstraints(link_sliced_center, dk_link_sliced_center, nullptr, values + (NUM_TIME_STEPS * NUM_FACTORS + 3*NUM_TIME_STEPS) * NUM_FACTORS);

        // Part 4. (position & velocity) state limit constraints
        desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + 3 * NUM_TIME_STEPS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles) * NUM_FACTORS, x);
        desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + 3 * NUM_TIME_STEPS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
    }

    // auto stop_g_grad = std::chrono::high_resolution_clock::now();
    // auto duration_g_grad = std::chrono::duration_cast<std::chrono::milliseconds>(stop_g_grad - start_g_grad);
    // cout << "        Time Taken to Calculate Eval g Grad Function: " << duration_g_grad.count() << " milliseconds" << endl;

    return true;
}
// [TNLP_eval_jac_g]


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (double)x[i];
    }

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    memcpy(g_copy, g, m * sizeof(Number));

    feasible = true;

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            if (g_copy[i * NUM_FACTORS + j] < -torque_limits[j] + (*torque_radius)(j, i) - TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD || 
                g_copy[i * NUM_FACTORS + j] > torque_limits[j] - (*torque_radius)(j, i) + TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD) {
                feasible = false;
                cout << "        CUDA & C++: Ipopt: Control torque of joint " << j << " at time interval " << i << " exceeds limit!\n";
                cout << "                        value: " << g_copy[i * NUM_FACTORS + j] << "\n";
                cout << "                        range: [ " << -torque_limits[j] + (*torque_radius)(j, i) << ", "
                                                            << torque_limits[j] - (*torque_radius)(j, i) << " ]\n";
                return;
            }
        }
    }    

    // collision avoidance constraints
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++) {
        // separation constraint
        // bool sep_constraint = (g_copy[i] > SEPARATION_CONSTRAINT_VIOLATION_THRESHOLD);
        // cout << sep_constraint << endl;
        if(g_copy[i] > SEPARATION_CONSTRAINT_VIOLATION_THRESHOLD) {
            feasible = false;
            double t_violation = i - offset;
            cout << "        CUDA & C++: Ipopt: Separation constraint violated at time interval: " << t_violation << " with value: " << g_copy[i] << "\n";
            return;
        }
        // slipping constraint
        if(g_copy[i+NUM_TIME_STEPS] > SLIPPING_CONSTRAINT_VIOLATION_THRESHOLD){
            feasible = false;
            double t_violation = i - offset;
            cout << "        CUDA & C++: Ipopt: Slipping constraint violated at time interval: " << t_violation << " with value: " << g_copy[i+NUM_TIME_STEPS] << " \n";
            return;
        }
        // tipping constraint
        if(g_copy[i+2*NUM_TIME_STEPS] > TIPPING_CONSTRAINT_VIOLATION_THRESHOLD){
            feasible = false;
            double t_violation = i - offset;
            cout << "        CUDA & C++: Ipopt: Tipping constraint violated at time interval: " << t_violation << " with value: " << g_copy[i+2*NUM_TIME_STEPS] << " \n";
            return;
        }
    }
    offset +=  NUM_TIME_STEPS*3;



    for( Index i = 0; i < NUM_JOINTS; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
