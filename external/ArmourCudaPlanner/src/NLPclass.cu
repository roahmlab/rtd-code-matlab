#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU

#include "NLPclass.h"

// constructor
armtd_NLP::armtd_NLP()
{
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    Eigen::VectorXd& q_des_input,
    double t_plan_input,
    const BezierCurve* desired_trajectory_input,
    KinematicsDynamics* kinematics_dynamics_result_input,
    const Eigen::MatrixXd* torque_radius_input,
    Obstacles* obstacles_input
 ) 
 {
    q_des = q_des_input;
    t_plan = t_plan_input;
    desired_trajectory = desired_trajectory_input;
    kinematics_dynamics_result = kinematics_dynamics_result_input;
    torque_radius = torque_radius_input;
    obstacles = obstacles_input;

    constraint_number = NUM_FACTORS * NUM_TIME_STEPS +
                        NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles + 
                        NUM_FACTORS * 4;

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            g_l[i * NUM_FACTORS + j] = -torque_limits[j] + (*torque_radius)(j, i);
            g_u[i * NUM_FACTORS + j] = torque_limits[j] - (*torque_radius)(j, i);
        }
    }    

    // collision avoidance constraints
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;
    for( Index i = offset; i < offset + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        x[i] = 0.0;

        // try to avoid local minimum
        // x[i] = min(max((q_des[i] - desired_trajectory->q0[i]) / k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // obj_value = sum((q_plan - q_des).^2);
    obj_value = 0; 
    for(Index i = 0; i < n; i++){
        double q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        obj_value += pow(q_plan - q_des[i], 2);
    }

    obj_value *= COST_FUNCTION_OPTIMALITY_SCALE;

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    for(Index i = 0; i < n; i++){
        double q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        double dk_q_plan = pow(t_plan,3) * (6 * pow(t_plan,2) - 15 * t_plan + 10);
        grad_f[i] = (2 * (q_plan - q_des[i]) * dk_q_plan * k_range[i]) * COST_FUNCTION_OPTIMALITY_SCALE;
    }

    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    Index i;
    #pragma omp parallel for shared(kinematics_dynamics_result, x, g, link_sliced_center) private(i) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
    for(i = 0; i < NUM_TIME_STEPS; i++) {
        for (int k = 0; k < NUM_FACTORS; k++) {
            MatrixXInt res = kinematics_dynamics_result->u_nom(k, i).slice(x);
            g[i * NUM_FACTORS + k] = getCenter(res(0));
        }

        for (int l = 0; l < NUM_JOINTS; l++) {
            MatrixXInt res = kinematics_dynamics_result->links(l, i).slice(x);
            link_sliced_center[i * NUM_JOINTS + l] = getCenter(res);
        }
    }

    // Part 3. check collision between joint position reachable set and obstacles (in gpu)
    obstacles->linkFRSConstraints(link_sliced_center, nullptr, g + NUM_TIME_STEPS * NUM_FACTORS, nullptr);

    // Part 4. (position & velocity) state limit constraints
    desired_trajectory->returnJointPositionExtremum(g + NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles, x);
    desired_trajectory->returnJointVelocityExtremum(g + NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2, x);

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }
        
    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;
        #pragma omp parallel for shared(kinematics_dynamics_result, x, values, link_sliced_center, dk_link_sliced_center) private(i) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(i = 0; i < NUM_TIME_STEPS; i++) {
            for (int k = 0; k < NUM_FACTORS; k++) {
                kinematics_dynamics_result->u_nom(k, i).slice(values + (i * NUM_FACTORS + k) * NUM_FACTORS, x);
            }

            for (int l = 0; l < NUM_JOINTS; l++) {
                link_sliced_center[i * NUM_JOINTS + l] = getCenter(kinematics_dynamics_result->links(l, i).slice(x));
                kinematics_dynamics_result->links(l, i).slice(dk_link_sliced_center + (i * NUM_JOINTS + l) * NUM_FACTORS, x);
            }
        }

        // Part 3. check collision between joint position reachable set and obstacles (in gpu)
        obstacles->linkFRSConstraints(link_sliced_center, dk_link_sliced_center, nullptr, values + NUM_TIME_STEPS * NUM_FACTORS * NUM_FACTORS);

        // Part 4. (position & velocity) state limit constraints
        desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles) * NUM_FACTORS, x);
        desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
    }

    return true;
}
// [TNLP_eval_jac_g]


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (double)x[i];
    }

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    memcpy(g_copy, g, m * sizeof(Number));

    feasible = true;

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            if (g_copy[i * NUM_FACTORS + j] < -torque_limits[j] + (*torque_radius)(j, i) - TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD || 
                g_copy[i * NUM_FACTORS + j] > torque_limits[j] - (*torque_radius)(j, i) + TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD) {
                feasible = false;
                cout << "        CUDA & C++: Ipopt: Control torque of joint " << j << " at time interval " << i << " exceeds limit!\n";
                cout << "                        value: " << g_copy[i * NUM_FACTORS + j] << "\n";
                cout << "                        range: [ " << -torque_limits[j] + (*torque_radius)(j, i) << ", "
                                                            << torque_limits[j] - (*torque_radius)(j, i) << " ]\n";
                return;
            }
        }
    }    

    // collision avoidance constraints
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;
    for( Index i = 0; i < NUM_JOINTS; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
