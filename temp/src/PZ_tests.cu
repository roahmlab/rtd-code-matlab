#include "hip/hip_runtime.h"
#include "Dynamics.h"
#include "CollisionChecking.h"

const std::string pathname = "/home/roahmlab/Documents/armour-dev/kinova_src/kinova_simulator_interfaces/kinova_planner_realtime/buffer/";
const std::string inputfilename = pathname + "armour.in";
const std::string outputfilename1 = pathname + "armour.out";
const std::string outputfilename2 = pathname + "armour_joint_position_center.out";
const std::string outputfilename3 = pathname + "armour_joint_position_radius.out";
const std::string outputfilename4 = pathname + "armour_control_input_radius.out";
const std::string outputfilename5 = pathname + "armour_constraints.out";

int main() {
/*
Section I:
    Parse input
    There is no check and warning, so be careful!
*/
    // Here is an example of the required input
    // double q0[NUM_FACTORS] = {0.6543, -0.0876, -0.4837, -1.2278, -1.5735, -1.0720, 0};
    // double qd0[NUM_FACTORS] = {0, 0, 0, 0, 0, 0, 0};
    // double qdd0[NUM_FACTORS] = {0, 0, 0, 0, 0, 0, 0};
    // double q_des[NUM_FACTORS] = {0.6831, 0.009488, -0.2471, -0.9777, -1.414, -0.9958, 0};

    // const int num_obstacles = 10;
    // const double obstacles[num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {-0.28239,  -0.33281, 0.88069, 0.069825, 0, 0, 0,  0.09508, 0, 0, 0, 0.016624,
    //                                                                             -0.19033,  0.035391,  1.3032,  0.11024, 0, 0, 0, 0.025188, 0, 0, 0, 0.014342,
    //                                                                             0.67593, -0.085841, 0.43572,  0.17408, 0, 0, 0,  0.07951, 0, 0, 0,  0.18012,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             -0.28239,  -0.33281, 0.88069, 0.069825, 0, 0, 0,  0.09508, 0, 0, 0, 0.016624,
    //                                                                             -0.19033,  0.035391,  1.3032,  0.11024, 0, 0, 0, 0.025188, 0, 0, 0, 0.014342,
    //                                                                             0.67593, -0.085841, 0.43572,  0.17408, 0, 0, 0,  0.07951, 0, 0, 0,  0.18012,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981,
    //                                                                             0.75382,   0.51895,  0.4731, 0.030969, 0, 0, 0,  0.22312, 0, 0, 0,  0.22981};

    // declare this first and make sure we always have a new output
    std::ofstream outputstream1(outputfilename1);

    Eigen::VectorXd q0(NUM_FACTORS); q0.setZero();
    Eigen::VectorXd qd0(NUM_FACTORS); qd0.setZero();
    Eigen::VectorXd qdd0(NUM_FACTORS); qdd0.setZero();
    Eigen::VectorXd q_des(NUM_FACTORS); q_des.setZero();

    int num_obstacles = 0;
    double obstacles[MAX_OBSTACLE_NUM * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {0.0};

    std::ifstream inputstream(inputfilename);
    if (!inputstream.is_open()) {
        WARNING_PRINT("        CUDA & C++: Error reading input files !\n");
        outputstream1 << -1;
        outputstream1.close();
        throw;
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> q0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> qd0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> qdd0[i];
    }
    for (int i = 0; i < NUM_FACTORS; i++) {
        inputstream >> q_des[i];
    }
    inputstream >> num_obstacles;
    if (num_obstacles > MAX_OBSTACLE_NUM || num_obstacles < 0) {
        WARNING_PRINT("        CUDA & C++: Number of obstacles larger than MAX_OBSTACLE_NUM !\n");
        outputstream1 << -1;
        outputstream1.close();
        throw;
    }
    if (num_obstacles > 0) {
        for (int i = 0; i < num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3; i++) {
            inputstream >> obstacles[i];
        }
    }

    inputstream.close();

    double t_plan = 1.0; // optimize the distance between q_des and the desired trajectories at t_plan
     
/*
Section II:
    Initialize all polynomial zonotopes, including links and torques
*/
    auto start1 = std::chrono::high_resolution_clock::now();

    omp_set_num_threads(NUM_THREADS);
    int openmp_t_ind = 0; // openmp loop index

    /*
    Section II.A: Create JRS online
    */
    BezierCurve traj(q0, qd0, qdd0);

    try {
        #pragma omp parallel for shared(traj) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
            traj.makePolyZono(openmp_t_ind);
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error creating JRS! Check previous error message!");
        return -1;
    }

    /*
    Section II.B: Compute link PZs and nominal torque PZs
    */
    KinematicsDynamics kd(&traj);
    Eigen::Matrix<double, 3, 3 + 3> link_independent_generators[NUM_TIME_STEPS * NUM_JOINTS];

    try {
        #pragma omp parallel for shared(kd, link_independent_generators) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
        for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
            // compute link PZs through forward kinematics
            kd.fk(openmp_t_ind);

            // reduce non-only-k-dependent generators so that slice takes less time
            for (int i = 0; i < NUM_JOINTS; i++) {
                link_independent_generators[openmp_t_ind * NUM_JOINTS + i] = kd.links(i, openmp_t_ind).reduce_link_PZ();
            }

            // compute nominal torque
            kd.rnea_nominal(openmp_t_ind);

            // compute interval torque
            kd.rnea_interval(openmp_t_ind);

            // compute max disturbance (stored in u_nom_int)
            for (int i = 0; i < NUM_FACTORS; i++) {
                kd.u_nom_int(i, openmp_t_ind) = kd.u_nom_int(i, openmp_t_ind) - kd.u_nom(i, openmp_t_ind);
            }

            // reduce non-only-k-dependent generators so that slice takes less time
            for (int i = 0; i < NUM_FACTORS; i++) {
                kd.u_nom(i, openmp_t_ind).reduce();
            }
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error computing link PZs and nominal torque PZs! Check previous error message!");
        return -1;
    }

    /*
    Section II.C: Compute robust input bound
    */
    // the radius of the torque PZs
    Eigen::MatrixXd torque_radius(NUM_FACTORS, NUM_TIME_STEPS);
    torque_radius.setZero();

    try {
        for(int t_ind = 0; t_ind < NUM_TIME_STEPS; t_ind++) {
            // (1) add the bound of robust input (||v||)
            Interval rho_max_temp = Interval(0.0);
            for (int i = 0; i < NUM_FACTORS; i++) {
                // compute norm of disturbance
                MatrixXInt temp = kd.u_nom_int(i, t_ind).toInterval(); // should be a 1-dim Interval
                rho_max_temp += temp(0) * temp(0);

                torque_radius(i, t_ind) = alpha * (M_max - M_min) * eps + 0.5 * max(abs(temp(0).lower()), abs(temp(0).upper()));
            }
            rho_max_temp = sqrt(rho_max_temp);
            
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += 0.5 * rho_max_temp.upper();
            }

            // (2) add the radius of the nominal input PZ (after reducing)
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += kd.u_nom(i, t_ind).independent(0);
            }

            // (3) add friction
            for (int i = 0; i < NUM_FACTORS; i++) {
                torque_radius(i, t_ind) += friction[i];
            }

            // so that torque_radius would be the radius of the total control input PZ from now
        }
    }
    catch (int errorCode) {
        WARNING_PRINT("        CUDA & C++: Error computing torque PZs! Check previous error message!");
        return -1;
    }

    auto stop1 = std::chrono::high_resolution_clock::now();
    auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start1);
    cout << "        CUDA & C++: Time taken by generating reachable sets: " << duration1.count() << " milliseconds" << endl;

/*
Section III:
    Slice reachable sets at some point
*/

    double factors[NUM_FACTORS] = {0.5, 0.6, 0.7, 0.0, -0.5, -0.6, -0.7};

    Eigen::MatrixXd qd_des_sliced_center(NUM_FACTORS, NUM_TIME_STEPS);
    Eigen::MatrixXd torque_sliced_center(NUM_FACTORS, NUM_TIME_STEPS);
    Eigen::Vector3d link_sliced_center[NUM_TIME_STEPS * NUM_JOINTS];

    #pragma omp parallel for shared(kd, factors, qd_des_sliced_center, torque_sliced_center, link_sliced_center) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
    for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
        for (int k = 0; k < NUM_FACTORS; k++) {
            MatrixXInt res = traj.qd_des(k, openmp_t_ind).slice(factors);
            qd_des_sliced_center(k, openmp_t_ind) = getCenter(res(0));
        }

        for (int k = 0; k < NUM_FACTORS; k++) {
            MatrixXInt res = kd.u_nom(k, openmp_t_ind).slice(factors);
            torque_sliced_center(k, openmp_t_ind) = getCenter(res(0));
        }

        for (int l = 0; l < NUM_JOINTS; l++) {
            MatrixXInt res = kd.links(l, openmp_t_ind).slice(factors);
            link_sliced_center[openmp_t_ind * NUM_JOINTS + l] = getCenter(res);
        }
    }

/*
Section IV:
    Prepare output
*/
    // output FRS and other information, you can comment them if they are unnecessary
    // std::ofstream outputstream2(outputfilename2);
    // outputstream2 << std::setprecision(10);
    // for (int i = 0; i < NUM_TIME_STEPS; i++) {
    //     for (int j = 0; j < NUM_JOINTS; j++) {
    //         for (int l = 0; l < 3; l++) {
    //             outputstream2 << link_sliced_center[i * NUM_JOINTS + j](l) << ' ';
    //         }
    //         outputstream2 << '\n';
    //     }
    //     outputstream2 << '\n';
    // }
    // outputstream2.close();
    std::ofstream outputstream2(outputfilename2);
    outputstream2 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_JOINTS; j++) {
            for (int l = 0; l < 3; l++) {
                outputstream2 << link_sliced_center[i * NUM_JOINTS + j](l) << ' ';
            }
            outputstream2 << '\n';
        }
        outputstream2 << '\n';
    }
    outputstream2.close();

    std::ofstream outputstream3(outputfilename3);
    outputstream3 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_JOINTS; j++) {
            for (int k = 0; k < 3; k++) {
                for (int l = 0; l < 3 + 3; l++) {
                    outputstream3 << link_independent_generators[i * NUM_JOINTS + j](k, l) << ' ';
                }
                outputstream3 << '\n';
            }
            outputstream3 << '\n';
        }
        outputstream3 << '\n';
    }
    outputstream3.close();

    std::ofstream outputstream4(outputfilename4);
    outputstream4 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_FACTORS; j++) {
            // outputstream4 << torque_radius(j, i) << ' '; // this is radius of final control input
            outputstream4 << kd.u_nom(j, i).independent(0) << ' '; // this is radius nominal torque
        }
        outputstream4 << '\n';
    }
    outputstream4.close();

    std::ofstream outputstream5(outputfilename5);
    outputstream5 << std::setprecision(10);
    for (int i = 0; i < NUM_TIME_STEPS; i++) {
        for (int j = 0; j < NUM_FACTORS; j++) {
            // outputstream5 << torque_sliced_center(j, i) << ' ';
            outputstream5 << qd_des_sliced_center(j, i) << ' ';
        }
        outputstream5 << '\n';
    }
    outputstream5.close();
}