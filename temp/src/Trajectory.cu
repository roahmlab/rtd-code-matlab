#include "hip/hip_runtime.h"
#ifndef TRAJECTORY_CPP
#define TRAJECTORY_CPP

#include "Trajectory.h"

BezierCurve::BezierCurve() {
    q0 = Eigen::VectorXd::Zero(NUM_FACTORS);
    qd0 = Eigen::VectorXd::Zero(NUM_FACTORS);
    qdd0 = Eigen::VectorXd::Zero(NUM_FACTORS);
    Tqd0 = Eigen::VectorXd::Zero(NUM_FACTORS);
    TTqdd0 = Eigen::VectorXd::Zero(NUM_FACTORS);
    ds = 1.0 / NUM_TIME_STEPS;
}

BezierCurve::BezierCurve(const Eigen::VectorXd& q0_inp, 
                         const Eigen::VectorXd& qd0_inp, 
                         const Eigen::VectorXd& qdd0_inp) {
    q0 = q0_inp;
    qd0 = qd0_inp;
    qdd0 = qdd0_inp;   

    Tqd0 = qd0 * DURATION; 
    TTqdd0 = qdd0 * DURATION * DURATION; 

    // pre-allocate memory
    R = PZsparseArray(NUM_JOINTS + 1, NUM_TIME_STEPS);
    R_t = PZsparseArray(NUM_JOINTS, NUM_TIME_STEPS);
    qd_des = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    qda_des = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    qdda_des = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);

    // initialize the extrema of the k independent part of q_des
    for (int i = 0; i < NUM_FACTORS; i++) {
        q_des_k_indep_extrema_1[i] = (2*Tqd0[i] + TTqdd0[i] + sqrt(64*pow(Tqd0[i],2) + 14*Tqd0[i]*TTqdd0[i] + pow(TTqdd0[i],2)))/(5*(6*Tqd0[i] + TTqdd0[i]));
        q_des_k_indep_extrema_2[i] = (2*Tqd0[i] + TTqdd0[i] - sqrt(64*pow(Tqd0[i],2) + 14*Tqd0[i]*TTqdd0[i] + pow(TTqdd0[i],2)))/(5*(6*Tqd0[i] + TTqdd0[i]));
        q_des_k_indep_extremum_1[i] = q_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], q_des_k_indep_extrema_1[i]);
        q_des_k_indep_extremum_2[i] = q_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], q_des_k_indep_extrema_2[i]);
    }

    // initialize the extrema of the k independent part of qd_des
    for (int i = 0; i < NUM_FACTORS; i++) {
        qd_des_k_indep_extrema_1[i] = (18*Tqd0[i] + 4*TTqdd0[i] + sqrt(6*(54*pow(Tqd0[i],2) + 14*Tqd0[i]*TTqdd0[i] + pow(TTqdd0[i],2))))/(10*(6*Tqd0[i] + TTqdd0[i]));
        qd_des_k_indep_extrema_2[i] = (18*Tqd0[i] + 4*TTqdd0[i] - sqrt(6*(54*pow(Tqd0[i],2) + 14*Tqd0[i]*TTqdd0[i] + pow(TTqdd0[i],2))))/(10*(6*Tqd0[i] + TTqdd0[i]));
        qd_des_k_indep_extremum_1[i] = qd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], qd_des_k_indep_extrema_1[i]);
        qd_des_k_indep_extremum_2[i] = qd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], qd_des_k_indep_extrema_2[i]);
    }

    // initialize the extrema of the k independent part of qdd_des
    for (int i = 0; i < NUM_FACTORS; i++) {
        qdd_des_k_indep_extrema_1[i] = (32*Tqd0[i] + 6*TTqdd0[i] + sqrt(2*(152*pow(Tqd0[i],2) + 42*Tqd0[i]*TTqdd0[i] + 3*pow(TTqdd0[i],2))))/(10*(6*Tqd0[i] + TTqdd0[i]));
        qdd_des_k_indep_extrema_2[i] = (32*Tqd0[i] + 6*TTqdd0[i] - sqrt(2*(152*pow(Tqd0[i],2) + 42*Tqd0[i]*TTqdd0[i] + 3*pow(TTqdd0[i],2))))/(10*(6*Tqd0[i] + TTqdd0[i]));
        qdd_des_k_indep_extremum_1[i] = qdd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], qdd_des_k_indep_extrema_1[i]);
        qdd_des_k_indep_extremum_2[i] = qdd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], qdd_des_k_indep_extrema_2[i]);
    }

    ds = 1.0 / NUM_TIME_STEPS;
}

void BezierCurve::makePolyZono(int s_ind) {
    assert(s_ind < NUM_TIME_STEPS);

    const double s_lb = s_ind * ds;
    const double s_ub = (s_ind + 1) * ds;

    const Interval t_int(s_lb, s_ub);

    for (int i = 0; i < NUM_FACTORS; i++) {
        const double k_range_elt = k_range[i];

        // Part 1: q_des
        double k_dep_coeff_lb = pow(s_lb,3) * (6 * pow(s_lb,2) - 15 * s_lb + 10);
        double k_dep_coeff_ub = pow(s_ub,3) * (6 * pow(s_ub,2) - 15 * s_ub + 10);
        double k_dep_coeff_center = (k_dep_coeff_ub + k_dep_coeff_lb) * 0.5;
        double k_dep_coeff_radius = (k_dep_coeff_ub - k_dep_coeff_lb) * 0.5 * k_range_elt;
        
        double k_indep_lb = q_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], s_lb);
        double k_indep_ub = q_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], s_ub);
        if (k_indep_lb > k_indep_ub) {
            swap(k_indep_lb, k_indep_ub);
        }
        if (s_lb < q_des_k_indep_extrema_1[i] && q_des_k_indep_extrema_1[i] < s_ub) {
            k_indep_lb = min(k_indep_lb, q_des_k_indep_extremum_1[i]);
            k_indep_ub = max(k_indep_ub, q_des_k_indep_extremum_1[i]);
        }
        if (s_lb < q_des_k_indep_extrema_2[i] && q_des_k_indep_extrema_2[i] < s_ub) {
            k_indep_lb = min(k_indep_lb, q_des_k_indep_extremum_2[i]);
            k_indep_ub = max(k_indep_ub, q_des_k_indep_extremum_2[i]);
        }
        double k_indep_radius = (k_indep_ub - k_indep_lb) * 0.5;
        double q_des_center = (k_indep_lb + k_indep_ub) * 0.5;
        
        // q_des_k_dep = k_dep_coeff_center * k;
        Interval q_des_radius_int(-k_dep_coeff_radius - k_indep_radius - qe, k_dep_coeff_radius + k_indep_radius + qe);
        
        // q_des_int = q_des_center + q_des_k_dep + q_des_radius_int;

        // first order Taylor expansion
        // Part 1.a: cos(q_des) 
        double cos_q_des_center = cos(q_des_center);
        Interval cos_q_des_radius_int = - q_des_radius_int * sin(q_des_center) 
                                        - 0.5 * cos(q_des_center + k_dep_coeff_center * Interval(-k_range_elt, k_range_elt) + q_des_radius_int) 
                                            * pow(q_des_radius_int + k_dep_coeff_center * Interval(-k_range_elt, k_range_elt), 2);

        cos_q_des_center += getCenter(cos_q_des_radius_int);
        cos_q_des_radius_int = cos_q_des_radius_int - getCenter(cos_q_des_radius_int);
        double cos_q_des_coeff[] = {-k_dep_coeff_center * k_range_elt * sin(q_des_center), getRadius(cos_q_des_radius_int)}; 

        // cos_q_des_int = cos_q_des_center + cos_q_des_coeff[0] * k + cos_q_des_coeff[1] * cosqe;
        uint64_t cos_q_des_degree[2][NUM_FACTORS * 6] = {0};
        cos_q_des_degree[0][i] = 1; // k
        cos_q_des_degree[1][i + NUM_FACTORS * 4] = 1; // cosqe

        // cos_q_des[s_ind * NUM_FACTORS + i] = PZsparse(cos_q_des_center, cos_q_des_coeff, cos_q_des_degree, 2);

        // Part 1.b: sin(q_des) 
        double sin_q_des_center = sin(q_des_center);
        Interval sin_q_des_radius_int = q_des_radius_int * cos(q_des_center) 
                                        - 0.5 * sin(q_des_center + k_dep_coeff_center * Interval(-k_range_elt, k_range_elt) + q_des_radius_int) 
                                            * pow(q_des_radius_int + k_dep_coeff_center * Interval(-k_range_elt, k_range_elt), 2);

        sin_q_des_center += getCenter(sin_q_des_radius_int);
        sin_q_des_radius_int = sin_q_des_radius_int - getCenter(sin_q_des_radius_int);
        double sin_q_des_coeff[] = {k_dep_coeff_center * k_range_elt * cos(q_des_center), getRadius(sin_q_des_radius_int)};

        // sin_q_des_int = sin_q_des_center + sin_q_des_coeff[0] * k + sin_q_des_coeff[1] * sinqe;
        uint64_t sin_q_des_degree[2][NUM_FACTORS * 6] = {0};
        sin_q_des_degree[0][i] = 1; // k
        sin_q_des_degree[1][i + NUM_FACTORS * 5] = 1; // sinqe

        // sin_q_des[s_ind * NUM_FACTORS + i] = PZsparse(sin_q_des_center, sin_q_des_coeff, sin_q_des_degree, 2);

        R(i, s_ind) = PZsparse(rots[i * 3], rots[i * 3 + 1], rots[i * 3 + 2]);

        if (axes[i] != 0) {
            R(i, s_ind) = R(i, s_ind) * PZsparse(cos_q_des_center, cos_q_des_coeff, cos_q_des_degree, 2,
                                                 sin_q_des_center, sin_q_des_coeff, sin_q_des_degree, 2,
                                                 axes[i]);
        }

        R_t(i, s_ind) = R(i, s_ind).transpose();
        
        // Part 2: qd_des
        // NOTE:
        // This is just a simplified implementation!!!
        // 30*t^2*(t - 1)^2 in qd_des is just a function with one maxima at t = 0.5
        // So as long as NUM_TIME_STEPS is even number, the following bounding trick holds!
        k_dep_coeff_lb = 30 * pow(s_lb,2) * pow(s_lb - 1,2);
        k_dep_coeff_ub = 30 * pow(s_ub,2) * pow(s_ub - 1,2);
        if (k_dep_coeff_ub < k_dep_coeff_lb) { // we are at t >= 0.5, which is a monotonically decreasing region 
            swap(k_dep_coeff_lb, k_dep_coeff_ub);
        }

        k_dep_coeff_center = (k_dep_coeff_ub + k_dep_coeff_lb) * 0.5 * k_range_elt; // Have to scale to [-1,1] in order to fit in PZsparse
        k_dep_coeff_radius = (k_dep_coeff_ub - k_dep_coeff_lb) * 0.5 * k_range_elt; // Have to scale to [-1,1] in order to fit in PZsparse

        k_indep_lb = qd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], s_lb);
        k_indep_ub = qd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], s_ub);
        if (k_indep_lb > k_indep_ub) {
            swap(k_indep_lb, k_indep_ub);
        }
        if (s_lb < qd_des_k_indep_extrema_1[i] && qd_des_k_indep_extrema_1[i] < s_ub) {
            k_indep_lb = min(k_indep_lb, qd_des_k_indep_extremum_1[i]);
            k_indep_ub = max(k_indep_ub, qd_des_k_indep_extremum_1[i]);
        }
        if (s_lb < qd_des_k_indep_extrema_2[i] && qd_des_k_indep_extrema_2[i] < s_ub) {
            k_indep_lb = min(k_indep_lb, qd_des_k_indep_extremum_2[i]);
            k_indep_ub = max(k_indep_ub, qd_des_k_indep_extremum_2[i]);
        }
        k_indep_radius = (k_indep_ub - k_indep_lb) * 0.5;
        double qd_des_center = (k_indep_lb + k_indep_ub) * 0.5;

        double qd_des_coeff[] = {k_dep_coeff_center, k_dep_coeff_radius + k_indep_radius + qde};

        uint64_t qd_des_degree[2][NUM_FACTORS * 6] = {0};
        qd_des_degree[0][i] = 1; // k
        qd_des_degree[1][i + NUM_FACTORS * 1] = 1; // qde

        // qd_des_int = qd_des_center + qd_des_coeff[0] * k + qd_des_coeff[1] * qde;
        qd_des(i, s_ind) = PZsparse(qd_des_center, qd_des_coeff, qd_des_degree, 2);

        double qda_des_coeff[] = {k_dep_coeff_center, k_dep_coeff_radius + k_indep_radius + qdae};

        uint64_t qda_des_degree[2][NUM_FACTORS * 6] = {0};
        qda_des_degree[0][i] = 1; // k
        qda_des_degree[1][i + NUM_FACTORS * 2] = 1; // qdae

        // qda_des_int = qd_des_center + qda_des_coeff[0] * k + qda_des_coeff[1] * qdae;
        qda_des(i, s_ind) = PZsparse(qd_des_center, qda_des_coeff, qda_des_degree, 2);

        // Part 3: qdd_des
        double temp_lb = 60 * s_lb * (2 * pow(s_lb,2) - 3 * s_lb + 1);
        double temp_ub = 60 * s_ub * (2 * pow(s_ub,2) - 3 * s_ub + 1);
        if (s_ub <= QDD_DES_K_DEP_MAXIMA) { // monotonically increasing region
            k_dep_coeff_lb = temp_lb;
            k_dep_coeff_ub = temp_ub;
        }
        else if (s_lb <= QDD_DES_K_DEP_MAXIMA) { // maxima lives inside
            k_dep_coeff_lb = min(temp_lb, temp_ub);
            k_dep_coeff_ub = 60 * QDD_DES_K_DEP_MAXIMA * (2 * pow(QDD_DES_K_DEP_MAXIMA,2) - 3 * QDD_DES_K_DEP_MAXIMA + 1);
        }
        else if (s_ub <= QDD_DES_K_DEP_MINIMA) { // monotonically decreasing region
            k_dep_coeff_lb = temp_ub;
            k_dep_coeff_ub = temp_lb;
        }
        else if (s_lb <= QDD_DES_K_DEP_MINIMA) { // minima lives inside
            k_dep_coeff_lb = 60 * QDD_DES_K_DEP_MINIMA * (2 * pow(QDD_DES_K_DEP_MINIMA,2) - 3 * QDD_DES_K_DEP_MINIMA + 1);
            k_dep_coeff_ub = max(temp_lb, temp_ub);
        }
        else { // monotonically increasing region
            k_dep_coeff_lb = temp_lb;
            k_dep_coeff_ub = temp_ub;
        }
        
        k_dep_coeff_center = (k_dep_coeff_ub + k_dep_coeff_lb) * 0.5 * k_range_elt; // Have to scale to [-1,1] in order to fit in PZsparse
        k_dep_coeff_radius = (k_dep_coeff_ub - k_dep_coeff_lb) * 0.5 * k_range_elt;

        k_indep_lb = qdd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], s_lb);
        k_indep_ub = qdd_des_k_indep(q0[i], Tqd0[i], TTqdd0[i], s_ub);
        if (k_indep_lb > k_indep_ub) {
            swap(k_indep_lb, k_indep_ub);
        }
        if (s_lb < qdd_des_k_indep_extrema_1[i] && qdd_des_k_indep_extrema_1[i] < s_ub) {
            k_indep_lb = min(k_indep_lb, qdd_des_k_indep_extremum_1[i]);
            k_indep_ub = max(k_indep_ub, qdd_des_k_indep_extremum_1[i]);
        }
        if (s_lb < qdd_des_k_indep_extrema_2[i] && qdd_des_k_indep_extrema_2[i] < s_ub) {
            k_indep_lb = min(k_indep_lb, qdd_des_k_indep_extremum_2[i]);
            k_indep_ub = max(k_indep_ub, qdd_des_k_indep_extremum_2[i]);
        }
        k_indep_radius = (k_indep_ub - k_indep_lb) * 0.5;
        double qdd_des_center = (k_indep_lb + k_indep_ub) * 0.5;

        double qdd_des_coeff[] = {k_dep_coeff_center, k_dep_coeff_radius + k_indep_radius + qddae};

        uint64_t qdd_des_degree[2][NUM_FACTORS * 6] = {0};
        qdd_des_degree[0][i] = 1; // k
        qdd_des_degree[1][i + NUM_FACTORS * 3] = 1; // qddae

        // qdd_des_int = qdd_des_center + qdd_des_coeff[0] * k + qdd_des_coeff[1] * qdde;
        qdda_des(i, s_ind) = PZsparse(qdd_des_center, qdd_des_coeff, qdd_des_degree, 2);
    }

    // assume all fixed joints are at the end of the kinematics chain
    for (int i = NUM_FACTORS; i < NUM_JOINTS; i++) {
        R(i, s_ind) = PZsparse(rots[i * 3], rots[i * 3 + 1], rots[i * 3 + 2]);
        R_t(i, s_ind) = R(i, s_ind).transpose();
    }

    R(NUM_JOINTS, s_ind) = PZsparse(0, 0, 0);
}

void BezierCurve::returnJointPositionExtremum(double* extremum, const double* k) const {
    for (int i = 0; i < NUM_FACTORS; i++) {
        // k[i] range is [-1,1] since it is defined for PZ, the following is the actual k
        double k_actual = k_range[i] * k[i];

        // list all possible extremas
        double extrema1 = 0;
        double extrema2 = (2*qd0[i] + qdd0[i] + sqrt(64*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] - 120*k_actual*qd0[i] + pow(qdd0[i],2))) / (5*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema3 = (2*qd0[i] + qdd0[i] - sqrt(64*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] - 120*k_actual*qd0[i] + pow(qdd0[i],2))) / (5*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema4 = 1;

        // get extremums of all extremas
        double extremum1 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema1);
        double extremum2 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema2);
        double extremum3 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema3);
        double extremum4 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema4);

        // find the min and max values
        double minPosition = min(extremum1, extremum4);
        double maxPosition = max(extremum1, extremum4);
        if (0 <= extrema2 && extrema2 <= 1) { // check if this extrema is inside the time range [0,1]
            minPosition = min(minPosition, extremum2);
            maxPosition = max(maxPosition, extremum2);
        }
        if (0 <= extrema3 && extrema3 <= 1) { // check if this extrema is inside the time range [0,1]
            minPosition = min(minPosition, extremum3);
            maxPosition = max(maxPosition, extremum3);
        }

        extremum[i              ] = minPosition;
        extremum[i + NUM_FACTORS] = maxPosition;
    }
}

void BezierCurve::returnJointPositionExtremumGradient(double* extremumGradient, const double* k) const {
    for (int i = 0; i < NUM_FACTORS; i++) {
        // k[i] range is [-1,1] since it is defined for PZ, the following is the actual k
        double k_actual = k_range[i] * k[i];

        // list all possible extremas
        double extrema1 = 0;
        double extrema2 = (2*qd0[i] + qdd0[i] + sqrt(64*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] - 120*k_actual*qd0[i] + pow(qdd0[i],2))) / (5*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema3 = (2*qd0[i] + qdd0[i] - sqrt(64*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] - 120*k_actual*qd0[i] + pow(qdd0[i],2))) / (5*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema4 = 1;

        // get extremums of all extremas
        double extremum1 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema1);
        double extremum2 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema2);
        double extremum3 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema3);
        double extremum4 = q_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema4);

        // find the min and max values
        double minPosition;
        int minId;
        double maxPosition;
        int maxId;

        if (extremum1 < extremum4) {
            minPosition = extremum1;
            minId = 1;

            maxPosition = extremum4;
            maxId = 4;
        }
        else {
            minPosition = extremum4;
            minId = 4;

            maxPosition = extremum1;
            maxId = 1;
        }

        if (0 <= extrema2 && extrema2 <= 1) { // check if this extrema is inside the time range [0,1]
            if (extremum2 < minPosition) {
                minPosition = extremum2;
                minId = 2;
            }
            if (maxPosition < extremum2) {
                maxPosition = extremum2;
                maxId = 2;
            }
        }
        if (0 <= extrema3 && extrema3 <= 1) { // check if this extrema is inside the time range [0,1]
            if (extremum3 < minPosition) {
                minPosition = extremum3;
                minId = 3;
            }
            if (maxPosition < extremum3) {
                maxPosition = extremum3;
                maxId = 3;
            }
        }

        double minPositionGradient;
        double maxPositionGradient;

        switch (minId) {
            case 1: // t = 0
                minPositionGradient = 0.0;
                break;
            case 2: // t = extrema2
                minPositionGradient = q_des_extrema2_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 3: // t = extrema3
                minPositionGradient = q_des_extrema3_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 4: // t = 1
                minPositionGradient = 1.0;
                break;
            default:
                break;
        }

        switch (maxId) {
            case 1: // t = 0
                maxPositionGradient = 0.0;
                break;
            case 2: // t = extrema2
                maxPositionGradient = q_des_extrema2_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 3: // t = extrema3
                maxPositionGradient = q_des_extrema3_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 4: // t = 1
                maxPositionGradient = 1.0;
                break;
            default:
                break;
        }

        for (int j = 0; j < NUM_FACTORS; j++) {
            if (i == j) {
                extremumGradient[(i              ) * NUM_FACTORS + j] = minPositionGradient * k_range[i];
                extremumGradient[(i + NUM_FACTORS) * NUM_FACTORS + j] = maxPositionGradient * k_range[i];
            }
            else {
                extremumGradient[(i              ) * NUM_FACTORS + j] = 0.0;
                extremumGradient[(i + NUM_FACTORS) * NUM_FACTORS + j] = 0.0;
            }
        }
    }
}

void BezierCurve::returnJointVelocityExtremum(double* extremum, const double* k) const {
    for (int i = 0; i < NUM_FACTORS; i++) {
        // k[i] range is [-1,1] since it is defined for PZ, the following is the actual k
        double k_actual = k_range[i] * k[i];

        // list all possible extremas
        double extrema1 = 0;
        double extrema2 = (18*qd0[i] - 30*k_actual + 4*qdd0[i] + sqrt(6*(150*pow(k_actual,2) - 180*k_actual*qd0[i] - 20*k_actual*qdd0[i] + 54*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] + pow(qdd0[i],2))))/(10*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema3 = (18*qd0[i] - 30*k_actual + 4*qdd0[i] - sqrt(6*(150*pow(k_actual,2) - 180*k_actual*qd0[i] - 20*k_actual*qdd0[i] + 54*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] + pow(qdd0[i],2))))/(10*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema4 = 1;

        // get extremums of all extremas
        double extremum1 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema1);
        double extremum2 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema2);
        double extremum3 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema3);
        double extremum4 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema4);

        // find the min and max values
        double minVelocity = min(extremum1, extremum4);
        double maxVelocity = max(extremum1, extremum4);
        if (0 <= extrema2 && extrema2 <= 1) { // check if this extrema is inside the time range [0,1]
            minVelocity = min(minVelocity, extremum2); 
            maxVelocity = max(maxVelocity, extremum2);
        }
        if (0 <= extrema3 && extrema3 <= 1) { // check if this extrema is inside the time range [0,1]
            minVelocity = min(minVelocity, extremum3);
            maxVelocity = max(maxVelocity, extremum3);
        }

        extremum[i              ] = minVelocity;
        extremum[i + NUM_FACTORS] = maxVelocity;
    }
}

void BezierCurve::returnJointVelocityExtremumGradient(double* extremumGradient, const double* k) const {
    for (int i = 0; i < NUM_FACTORS; i++) {
        // k[i] range is [-1,1] since it is defined for PZ, the following is the actual k
        double k_actual = k_range[i] * k[i];

        // list all possible extremas
        double extrema1 = 0;
        double extrema2 = (18*qd0[i] - 30*k_actual + 4*qdd0[i] + sqrt(6*(150*pow(k_actual,2) - 180*k_actual*qd0[i] - 20*k_actual*qdd0[i] + 54*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] + pow(qdd0[i],2))))/(10*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema3 = (18*qd0[i] - 30*k_actual + 4*qdd0[i] - sqrt(6*(150*pow(k_actual,2) - 180*k_actual*qd0[i] - 20*k_actual*qdd0[i] + 54*pow(qd0[i],2) + 14*qd0[i]*qdd0[i] + pow(qdd0[i],2))))/(10*(6*qd0[i] - 12*k_actual + qdd0[i]));
        double extrema4 = 1;

        // get extremums of all extremas
        double extremum1 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema1);
        double extremum2 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema2);
        double extremum3 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema3);
        double extremum4 = qd_des_func(q0[i], qd0[i], qdd0[i], k_actual, extrema4);

        // find the min and max values
        double minVelocity;
        int minId;
        double maxVelocity;
        int maxId;

        if (extremum1 < extremum4) {
            minVelocity = extremum1;
            minId = 1;

            maxVelocity = extremum4;
            maxId = 4;
        }
        else {
            minVelocity = extremum4;
            minId = 4;

            maxVelocity = extremum1;
            maxId = 1;
        }

        if (0 <= extrema2 && extrema2 <= 1) { // check if this extrema is inside the time range [0,1]
            if (extremum2 < minVelocity) {
                minVelocity = extremum2;
                minId = 2;
            }
            if (maxVelocity < extremum2) {
                maxVelocity = extremum2;
                maxId = 2;
            }
        }
        if (0 <= extrema3 && extrema3 <= 1) { // check if this extrema is inside the time range [0,1]
            if (extremum3 < minVelocity) {
                minVelocity = extremum3;
                minId = 3;
            }
            if (maxVelocity < extremum3) {
                maxVelocity = extremum3;
                maxId = 3;
            }
        }

        double minVelocityGradient;
        double maxVelocityGradient;

        switch (minId) {
            case 1: // t = 0
                minVelocityGradient = 0.0;
                break;
            case 2: // t = extrema2
                minVelocityGradient = qd_des_extrema2_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 3: // t = extrema3
                minVelocityGradient = qd_des_extrema3_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 4: // t = 1
                minVelocityGradient = 1.0;
                break;
            default:
                break;
        }

        switch (maxId) {
            case 1: // t = 0
                maxVelocityGradient = 0.0;
                break;
            case 2: // t = extrema2
                maxVelocityGradient = qd_des_extrema2_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 3: // t = extrema3
                maxVelocityGradient = qd_des_extrema3_k_derivative(q0[i], qd0[i], qdd0[i], k_actual);
                break;
            case 4: // t = 1
                maxVelocityGradient = 1.0;
                break;
            default:
                break;
        }

        for (int j = 0; j < NUM_FACTORS; j++) {
            if (i == j) {
                extremumGradient[(i              ) * NUM_FACTORS + j] = minVelocityGradient * k_range[i];
                extremumGradient[(i + NUM_FACTORS) * NUM_FACTORS + j] = maxVelocityGradient * k_range[i];
            }
            else {
                extremumGradient[(i              ) * NUM_FACTORS + j] = 0.0;
                extremumGradient[(i + NUM_FACTORS) * NUM_FACTORS + j] = 0.0;
            }
        }
    }
}

double q_des_func(double q0, double qd0, double qdd0, double k, double t) {
    double B0 = -pow(t - 1,5);
    double B1 = 5*t*pow(t - 1,4);
    double B2 = -10*pow(t,2)*pow(t - 1,3);
    double B3 = 10*pow(t,3)*pow(t - 1,2);
    double B4 = -5*pow(t,4)*(t - 1);
    double B5 = pow(t,5);
    double beta0 = q0;
    double beta1 = q0 + qd0/5;
    double beta2 = q0 + (2*qd0)/5 + qdd0/20;
    double beta3 = q0 + k;
    double beta4 = q0 + k;
    double beta5 = q0 + k;
    return B0 * beta0 + B1 * beta1 + B2 * beta2 + B3 * beta3 + B4 * beta4 + B5 * beta5;
}

double qd_des_func(double q0, double qd0, double qdd0, double k, double t) {
    double dB0 = pow(t-1.0,4.0)*-5.0;
    double dB1 = t*pow(t-1.0,3.0)*2.0E+1+pow(t-1.0,4.0)*5.0;
    double dB2 = t*pow(t-1.0,3.0)*-2.0E+1-(t*t)*pow(t-1.0,2.0)*3.0E+1;
    double dB3 = pow(t,3.0)*(t*2.0-2.0)*1.0E+1+(t*t)*pow(t-1.0,2.0)*3.0E+1;
    double dB4 = pow(t,3.0)*(t-1.0)*-2.0E+1-pow(t,4.0)*5.0;
    double dB5 = pow(t,4.0)*5.0;
    double beta0 = q0;
    double beta1 = q0 + qd0/5;
    double beta2 = q0 + (2*qd0)/5 + qdd0/20;
    double beta3 = q0 + k;
    double beta4 = q0 + k;
    double beta5 = q0 + k;
    return dB0 * beta0 + dB1 * beta1 + dB2 * beta2 + dB3 * beta3 + dB4 * beta4 + dB5 * beta5;
}

double qdd_des_func(double q0, double qd0, double qdd0, double k, double t) {
    double t2 = t*2.0;
    double t3 = t*t;
    double t4 = t*t*t;
    double t5 = t-1.0;
    double t6 = t2-2.0;
    double t7 = t4*2.0E+1;
    double t8 = t5*t5;
    double t9 = t5*t5*t5;
    double t10 = t9*2.0E+1;
    double t11 = t*t8*6.0E+1;
    double t12 = -t10;
    double ddB0 = t12;
    double ddB1 = t9*4.0E+1+t11;
    double ddB2 = t12-t*t8*1.2E+2-t3*t6*3.0E+1;
    double ddB3 = t7+t11+t3*t6*6.0E+1;
    double ddB4 = t4*-4.0E+1-t3*t5*6.0E+1;
    double ddB5 = t7;
    double beta0 = q0;
    double beta1 = q0 + qd0/5;
    double beta2 = q0 + (2*qd0)/5 + qdd0/20;
    double beta3 = q0 + k;
    double beta4 = q0 + k;
    double beta5 = q0 + k;
    return ddB0 * beta0 + ddB1 * beta1 + ddB2 * beta2 + ddB3 * beta3 + ddB4 * beta4 + ddB5 * beta5;
}

double q_des_extrema2_k_derivative(double q0, double qd0, double qdd0, double k) {
    double t2 = k+q0;
    double t3 = qd0*2.0;
    double t4 = qd0*6.0;
    double t5 = qd0*qd0;
    double t6 = qdd0*qdd0;
    double t7 = k*1.2E+1;
    double t8 = qd0*qdd0*1.4E+1;
    double t10 = qd0/5.0;
    double t11 = qd0*(2.0/5.0);
    double t12 = k*qd0*1.2E+2;
    double t13 = qdd0/2.0E+1;
    double t9 = -t7;
    double t14 = t5*6.4E+1;
    double t15 = -t12;
    double t16 = q0+t10;
    double t18 = q0+t11+t13;
    double t17 = qdd0+t4+t9;
    double t24 = t6+t8+t14+t15;
    double t19 = 1.0/t17;
    double t25 = sqrt(t24);
    double t20 = t19*t19;
    double t21 = t19*t19*t19;
    double t23 = t19*t19*t19*t19*t19;
    double t26 = 1.0/t25;
    double t27 = qdd0+t3+t25;
    double t22 = t20*t20;
    double t28 = t27*t27;
    double t29 = t27*t27*t27;
    double t31 = t27*t27*t27*t27*t27;
    double t32 = qd0*t19*t26*1.2E+1;
    double t34 = (t19*t27)/5.0;
    double t35 = t20*t27*(1.2E+1/5.0);
    double t30 = t28*t28;
    double t33 = -t32;
    double t36 = t34-1.0;
    double t37 = t36*t36;
    double t38 = t36*t36*t36;
    double t40 = t33+t35;
    double t39 = t37*t37;
    return (t23*t31)/3.125E+3+t2*(t20*t20*t20)*t31*(1.2E+1/6.25E+2)+t21*t29*t37*(2.0/2.5E+1)-(t22*t30*t36)/1.25E+2+q0*t39*(t32-t35)*5.0+t2*t22*t29*t37*(7.2E+1/2.5E+1)-t2*t23*t30*t36*(4.8E+1/1.25E+2)+t16*t20*t27*t39*1.2E+1-t18*t21*t28*t38*(4.8E+1/5.0)+(t2*t22*t30*(t32-t35))/1.25E+2-qd0*t2*t23*t26*t30*(1.2E+1/1.25E+2)-qd0*t16*t19*t26*t39*6.0E+1-t2*t21*t29*t36*(t32-t35)*(4.0/2.5E+1)-t16*t19*t27*t38*(t32-t35)*4.0+t18*t20*t28*t37*(t32-t35)*(6.0/5.0)-qd0*t2*t21*t26*t28*t37*(7.2E+1/5.0)+qd0*t2*t22*t26*t29*t36*(4.8E+1/2.5E+1)+qd0*t18*t20*t26*t27*t38*4.8E+1;
}

double q_des_extrema3_k_derivative(double q0, double qd0, double qdd0, double k) {
    double t2 = k+q0;
    double t3 = qd0*2.0;
    double t4 = qd0*6.0;
    double t5 = qd0*qd0;
    double t6 = qdd0*qdd0;
    double t7 = k*1.2E+1;
    double t8 = qd0*qdd0*1.4E+1;
    double t10 = qd0/5.0;
    double t11 = qd0*(2.0/5.0);
    double t12 = k*qd0*1.2E+2;
    double t13 = qdd0/2.0E+1;
    double t9 = -t7;
    double t14 = t5*6.4E+1;
    double t15 = -t12;
    double t16 = q0+t10;
    double t18 = q0+t11+t13;
    double t17 = qdd0+t4+t9;
    double t24 = t6+t8+t14+t15;
    double t19 = 1.0/t17;
    double t25 = sqrt(t24);
    double t20 = t19*t19;
    double t21 = t19*t19*t19;
    double t23 = t19*t19*t19*t19*t19;
    double t26 = 1.0/t25;
    double t27 = -t25;
    double t22 = t20*t20;
    double t28 = qdd0+t3+t27;
    double t33 = qd0*t19*t26*1.2E+1;
    double t29 = t28*t28;
    double t30 = t28*t28*t28;
    double t32 = t28*t28*t28*t28*t28;
    double t34 = (t19*t28)/5.0;
    double t35 = t20*t28*(1.2E+1/5.0);
    double t31 = t29*t29;
    double t36 = t34-1.0;
    double t40 = t33+t35;
    double t37 = t36*t36;
    double t38 = t36*t36*t36;
    double t39 = t37*t37;
    return (t23*t32)/3.125E+3+t2*(t20*t20*t20)*t32*(1.2E+1/6.25E+2)-q0*t39*t40*5.0+t21*t30*t37*(2.0/2.5E+1)-(t22*t31*t36)/1.25E+2+t2*t22*t30*t37*(7.2E+1/2.5E+1)-t2*t23*t31*t36*(4.8E+1/1.25E+2)-(t2*t22*t31*t40)/1.25E+2+t16*t20*t28*t39*1.2E+1-t18*t21*t29*t38*(4.8E+1/5.0)+qd0*t2*t23*t26*t31*(1.2E+1/1.25E+2)+qd0*t16*t19*t26*t39*6.0E+1+t2*t21*t30*t36*t40*(4.0/2.5E+1)+t16*t19*t28*t38*t40*4.0-t18*t20*t29*t37*t40*(6.0/5.0)+qd0*t2*t21*t26*t29*t37*(7.2E+1/5.0)-qd0*t2*t22*t26*t30*t36*(4.8E+1/2.5E+1)-qd0*t18*t20*t26*t28*t38*4.8E+1;
}

double qd_des_extrema2_k_derivative(double q0, double qd0, double qdd0, double k) {
    double t2 = k+q0;
    double t3 = k*k;
    double t4 = qd0*6.0;
    double t5 = qdd0*4.0;
    double t6 = qd0*qd0;
    double t7 = qdd0*qdd0;
    double t8 = k*1.2E+1;
    double t9 = k*3.0E+1;
    double t10 = qd0*1.8E+1;
    double t11 = qdd0*2.0E+1;
    double t13 = qd0*qdd0*1.4E+1;
    double t14 = sqrt(6.0);
    double t17 = k*3.0E+2;
    double t18 = qd0*1.8E+2;
    double t21 = k*qdd0*-2.0E+1;
    double t24 = k*qd0*-1.8E+2;
    double t12 = k*t11;
    double t15 = -t8;
    double t16 = -t9;
    double t19 = t6*5.4E+1;
    double t20 = k*t18;
    double t22 = -t17;
    double t23 = t3*1.5E+2;
    double t25 = qdd0+t4+t15;
    double t31 = t11+t18+t22;
    double t32 = t7+t13+t19+t21+t23+t24;
    double t26 = 1.0/t25;
    double t33 = sqrt(t32);
    double t27 = t26*t26;
    double t28 = t26*t26*t26;
    double t30 = t26*t26*t26*t26*t26;
    double t34 = 1.0/t33;
    double t35 = t14*t33;
    double t29 = t27*t27;
    double t36 = t5+t10+t16+t35;
    double t40 = (t14*t31*t34)/2.0;
    double t37 = t36*t36;
    double t38 = t36*t36*t36;
    double t41 = t40+3.0E+1;
    double t42 = (t26*t36)/5.0;
    double t43 = t27*t36*(6.0/5.0);
    double t44 = (t26*t36)/1.0E+1;
    double t39 = t37*t37;
    double t45 = t42-2.0;
    double t46 = t44-1.0;
    double t49 = (t26*t41)/1.0E+1;
    double t47 = t46*t46;
    double t48 = t46*t46*t46;
    double t50 = -t49;
    double t51 = t27*t36*t48*2.4E+1;
    double t52 = t28*t37*t47*(3.6E+1/5.0);
    double t53 = t43+t50;
    double t54 = t26*t41*t48*2.0;
    double t56 = t27*t36*t41*t47*(3.0/5.0);
    double t55 = -t54;
    double t57 = -t56;
    double t58 = t26*t36*t47*t53*6.0;
    double t59 = t27*t37*t46*t53*(3.0/5.0);
    return (q0+qd0/5.0)*(t51+t55+t58+t48*t53*2.0E+1)+t2*(t52+t57+t59+(t28*t38*(t27*t36*(1.2E+1/5.0)-(t26*t41)/5.0))/1.0E+2+t29*t38*t45*(9.0/2.5E+1)-t28*t37*t41*t45*(3.0/1.0E+2))-t2*(t30*t39*(3.0/1.25E+2)-(t29*t38*t41)/5.0E+2+t29*t38*t46*(1.8E+1/2.5E+1)+(t28*t38*t53)/5.0E+1-t28*t37*t41*t46*(3.0/5.0E+1))-(q0+qd0*(2.0/5.0)+qdd0/2.0E+1)*(t51+t52+t55+t57+t58+t59)-q0*t48*t53*2.0E+1+t2*t30*t39*(3.0/1.25E+2)+t27*t37*t47*(3.0/1.0E+1)+(t28*t38*t45)/1.0E+2-(t28*t38*t46)/5.0E+1-(t2*t29*t38*t41)/5.0E+2;
}

double qd_des_extrema3_k_derivative(double q0, double qd0, double qdd0, double k) {
    double t2 = k+q0;
    double t3 = k*k;
    double t4 = qd0*6.0;
    double t5 = qdd0*4.0;
    double t6 = qd0*qd0;
    double t7 = qdd0*qdd0;
    double t8 = k*1.2E+1;
    double t9 = k*3.0E+1;
    double t10 = qd0*1.8E+1;
    double t12 = qdd0*2.0E+1;
    double t14 = qd0*qdd0*1.4E+1;
    double t15 = sqrt(6.0);
    double t17 = k*3.0E+2;
    double t19 = qd0*1.8E+2;
    double t22 = k*qdd0*-2.0E+1;
    double t25 = k*qd0*-1.8E+2;
    double t11 = -t5;
    double t13 = k*t12;
    double t16 = -t8;
    double t18 = -t10;
    double t20 = t6*5.4E+1;
    double t21 = k*t19;
    double t23 = -t17;
    double t24 = t3*1.5E+2;
    double t26 = qdd0+t4+t16;
    double t32 = t12+t19+t23;
    double t33 = t7+t14+t20+t22+t24+t25;
    double t27 = 1.0/t26;
    double t34 = sqrt(t33);
    double t28 = t27*t27;
    double t29 = t27*t27*t27;
    double t31 = t27*t27*t27*t27*t27;
    double t35 = 1.0/t34;
    double t36 = t15*t34;
    double t30 = t28*t28;
    double t37 = t9+t11+t18+t36;
    double t38 = pow(t5-t9+t10-t36,2.0);
    double t39 = -pow(t5-t9+t10-t36,3.0);
    double t41 = (t15*t32*t35)/2.0;
    double t43 = t27*(t5-t9+t10-t36)*(-1.0/5.0);
    double t44 = t28*(t5-t9+t10-t36)*(-6.0/5.0);
    double t45 = t27*(t5-t9+t10-t36)*(-1.0/1.0E+1);
    double t48 = pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,2.0);
    double t49 = -pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,3.0);
    double t52 = t28*pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,3.0)*(t5-t9+t10-t36)*2.4E+1;
    double t40 = t38*t38;
    double t42 = t41-3.0E+1;
    double t46 = t43+2.0;
    double t47 = t45+1.0;
    double t53 = t29*t38*t48*(3.6E+1/5.0);
    double t50 = (t27*t42)/1.0E+1;
    double t55 = t27*t42*pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,3.0)*-2.0;
    double t56 = t27*t42*pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,3.0)*2.0;
    double t57 = t28*t42*t48*(t5-t9+t10-t36)*(-3.0/5.0);
    double t58 = t28*t42*t48*(t5-t9+t10-t36)*(3.0/5.0);
    double t51 = -t50;
    double t59 = t27*t48*(t50+t28*(t5-t9+t10-t36)*(6.0/5.0))*(t5-t9+t10-t36)*6.0;
    double t60 = t28*t38*((t27*(t5-t9+t10-t36))/1.0E+1-1.0)*(t50+t28*(t5-t9+t10-t36)*(6.0/5.0))*(3.0/5.0);
    double t54 = t44+t51;
    return (q0+qd0/5.0)*(t52+t56+t59+pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,3.0)*(t50+t28*(t5-t9+t10-t36)*(6.0/5.0))*2.0E+1)+t2*(t53+t58+t60+t30*((t27*(t5-t9+t10-t36))/5.0-2.0)*pow(t5-t9+t10-t36,3.0)*(9.0/2.5E+1)+(t29*((t27*t42)/5.0+t28*(t5-t9+t10-t36)*(1.2E+1/5.0))*pow(t5-t9+t10-t36,3.0))/1.0E+2+t29*t38*t42*((t27*(t5-t9+t10-t36))/5.0-2.0)*(3.0/1.0E+2))-t2*(t31*t40*(3.0/1.25E+2)+t30*((t27*(t5-t9+t10-t36))/1.0E+1-1.0)*pow(t5-t9+t10-t36,3.0)*(1.8E+1/2.5E+1)+(t30*t42*pow(t5-t9+t10-t36,3.0))/5.0E+2+(t29*(t50+t28*(t5-t9+t10-t36)*(6.0/5.0))*pow(t5-t9+t10-t36,3.0))/5.0E+1+t29*t38*t42*((t27*(t5-t9+t10-t36))/1.0E+1-1.0)*(3.0/5.0E+1))-(q0+qd0*(2.0/5.0)+qdd0/2.0E+1)*(t52+t53+t56+t58+t59+t60)+(t29*((t27*(t5-t9+t10-t36))/5.0-2.0)*pow(t5-t9+t10-t36,3.0))/1.0E+2-(t29*((t27*(t5-t9+t10-t36))/1.0E+1-1.0)*pow(t5-t9+t10-t36,3.0))/5.0E+1+t2*t31*t40*(3.0/1.25E+2)+t28*t38*t48*(3.0/1.0E+1)-q0*pow((t27*(t5-t9+t10-t36))/1.0E+1-1.0,3.0)*(t50+t28*(t5-t9+t10-t36)*(6.0/5.0))*2.0E+1+(t2*t30*t42*pow(t5-t9+t10-t36,3.0))/5.0E+2;
}

double q_des_k_indep(double q0, double Tqd0, double TTqdd0, double s) {
    return q0 + Tqd0*s - 6*Tqd0*pow(s,3) + 8*Tqd0*pow(s,4) - 3*Tqd0*pow(s,5) + (TTqdd0*pow(s,2))*0.5 - (3*TTqdd0*pow(s,3))*0.5 + (3*TTqdd0*pow(s,4))*0.5 - (TTqdd0*pow(s,5))*0.5;
}

double qd_des_k_indep(double q0, double Tqd0, double TTqdd0, double s) {
    return (pow(s - 1,2)*(2*Tqd0 + 4*Tqd0*s + 2*TTqdd0*s - 30*Tqd0*pow(s,2) - 5*TTqdd0*pow(s,2)))*0.5 / DURATION;
}

double qdd_des_k_indep(double q0, double Tqd0, double TTqdd0, double s) {
    return -(s - 1.0)*(TTqdd0 - (36*Tqd0 + 8*TTqdd0)*s + (60*Tqd0 + 10*TTqdd0)*pow(s, 2))  / (DURATION * DURATION);
}

#endif