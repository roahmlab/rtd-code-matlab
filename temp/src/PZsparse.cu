#include "hip/hip_runtime.h"
#ifndef PZ_SPARSE_CPP
#define PZ_SPARSE_CPP

#include "PZsparse.h"

/*
Helper functions
*/

double getCenter(const Interval& a) {
    return (a.lower() + a.upper()) * 0.5;
}

double getRadius(const Interval& a) {
    return (a.upper() - a.lower()) * 0.5;
}

Eigen::MatrixXd getCenter(const MatrixXInt& a) {
    Eigen::MatrixXd res(a.rows(), a.cols());

    for (uint i = 0; i < a.rows(); i++) {
        for (uint j = 0; j < a.cols(); j++) {
            res(i, j) = getCenter(a(i, j));
        }
    }

    return res;
}

Eigen::MatrixXd getRadius(const MatrixXInt& a) {
    Eigen::MatrixXd res(a.rows(), a.cols());

    for (uint i = 0; i < a.rows(); i++) {
        for (uint j = 0; j < a.cols(); j++) {
            res(i, j) = getRadius(a(i, j));
        }
    }

    return res;
}

bool Monomial_sorter_degree(Monomial const& lhs, Monomial const& rhs) {
    return lhs.degree < rhs.degree;
}

/*
Initialization
*/

PZsparse::PZsparse(uint NRows_inp, uint NCols_inp) {
    NRows = NRows_inp;
    NCols = NCols_inp;
    center = Eigen::MatrixXd::Zero(NRows, NCols);
    independent = Eigen::MatrixXd::Zero(NRows, NCols);
}

PZsparse::PZsparse(const PZsparse& pz_inp) {
    NRows = pz_inp.NRows;
    NCols = pz_inp.NCols;
    center = pz_inp.center;
    polynomial = pz_inp.polynomial;
    independent = pz_inp.independent;
}

// 1x1 PZ
PZsparse::PZsparse(double center_inp) {
    NRows = 1;
    NCols = 1;
    center.resize(NRows, NCols);
    center(0) = center_inp;
    independent = Eigen::MatrixXd::Zero(NRows, NCols);
}

// NxM PZ
PZsparse::PZsparse(const Eigen::MatrixXd& center_inp) {
    NRows = center_inp.rows();
    NCols = center_inp.cols();
    center = center_inp;
    independent = Eigen::MatrixXd::Zero(NRows, NCols);
}

// 1x1 PZ
// PZsparse::PZsparse(double center_inp, double uncertainty_percent) {
//     NRows = 1;
//     NCols = 1;
//     center.resize(NRows, NCols);
//     center(0) = center_inp;
//     independent.resize(NRows, NCols);
//     independent(0) = uncertainty_percent * fabs(center_inp);
// }

// NxM PZ
PZsparse::PZsparse(const Eigen::MatrixXd& center_inp, double uncertainty_percent) {
    NRows = center_inp.rows();
    NCols = center_inp.cols();
    center = center_inp;
    independent = uncertainty_percent * center_inp.cwiseAbs();
}

// // 1x1 PZ
// PZsparse::PZsparse(Interval interval_inp) {
//     NRows = 1;
//     NCols = 1;
//     center.resize(NRows, NCols);
//     center(0) = getCenter(interval_inp);
//     independent.resize(NRows, NCols);
//     independent(0) = getRadius(interval_inp);
// }

PZsparse::PZsparse(double center_inp, Interval independent_inp) {
    NRows = 1;
    NCols = 1;
    center.resize(NRows, NCols);
    center(0) = center_inp + getCenter(independent_inp);
    independent.resize(NRows, NCols);
    independent(0) = getRadius(independent_inp);
}

// 1x1 PZ
PZsparse::PZsparse(double center_inp, double* coeff_inp, uint64_t degree_inp[][NUM_FACTORS * 6], uint num_monomials) {
    NRows = 1;
    NCols = 1;
    
    center.resize(NRows, NCols);
    center(0) = center_inp;

    polynomial.reserve(num_monomials);

    for (uint i = 0; i < num_monomials; i++) {
        polynomial.emplace_back(coeff_inp[i], convertDegreeToHash(degree_inp[i]));
    }

    independent = Eigen::MatrixXd::Zero(NRows, NCols);

    simplify();
}

// 1x1 PZ
PZsparse::PZsparse(double center_inp, double* coeff_inp, uint64_t degree_inp[][NUM_FACTORS * 6], uint num_monomials, Interval independent_inp) {
    NRows = 1;
    NCols = 1;
    
    center.resize(NRows, NCols);
    center(0) = center_inp;

    polynomial.reserve(num_monomials);

    for (uint i = 0; i < num_monomials; i++) {
        polynomial.emplace_back(coeff_inp[i], convertDegreeToHash(degree_inp[i]));
    }

    // assume independent_inp is centered at 0
    independent.resize(NRows, NCols);
    independent(0) = getRadius(independent_inp);

    simplify();
}

// 3x3 PZ
PZsparse::PZsparse(const double roll, const double pitch, const double yaw) {
    NRows = 3;
    NCols = 3;
    center.resize(NRows, NCols);
    
    center(0,0) = cos(pitch)*cos(yaw);
    center(0,1) = -cos(pitch)*sin(yaw);
    center(0,2) = sin(pitch);
    center(1,0) = cos(roll)*sin(yaw) + cos(yaw)*sin(pitch)*sin(roll);
    center(1,1) = cos(roll)*cos(yaw) - sin(pitch)*sin(roll)*sin(yaw);
    center(1,2) = -cos(pitch)*sin(roll);
    center(2,0) = sin(roll)*sin(yaw) - cos(roll)*cos(yaw)*sin(pitch);
    center(2,1) = cos(yaw)*sin(roll) + cos(roll)*sin(pitch)*sin(yaw);
    center(2,2) = cos(pitch)*cos(roll);

    independent = Eigen::MatrixXd::Zero(NRows, NCols);
}

// 3x3 PZ
PZsparse::PZsparse(double cos_center_inp, double* cos_coeff_inp, uint64_t cos_degree_inp[][NUM_FACTORS * 6], uint cos_num_monomials,
                   double sin_center_inp, double* sin_coeff_inp, uint64_t sin_degree_inp[][NUM_FACTORS * 6], uint sin_num_monomials,
                   const uint axis) {
    NRows = 3;
    NCols = 3;
    
    makeRotationMatrix(center, cos_center_inp, sin_center_inp, axis);

    polynomial.reserve(cos_num_monomials + sin_num_monomials);

    Eigen::MatrixXd coeff_temp;
    for (uint i = 0; i < cos_num_monomials; i++) {
        makeRotationMatrix(coeff_temp, cos_coeff_inp[i], 0, axis, true);
        polynomial.emplace_back(coeff_temp, convertDegreeToHash(cos_degree_inp[i]));
    }

    for (uint i = 0; i < sin_num_monomials; i++) {
        makeRotationMatrix(coeff_temp, 0, sin_coeff_inp[i], axis, true);
        polynomial.emplace_back(coeff_temp, convertDegreeToHash(sin_degree_inp[i]));
    }

    // assume independent_inp is centered at 0
    // makeRotationMatrix(independent, getRadius(cos_independent_inp), getRadius(sin_independent_inp), axis);
    independent = Eigen::MatrixXd::Zero(3, 3);

    simplify();
}

/*
Internal functions
*/

void PZsparse::makeRotationMatrix(Eigen::MatrixXd& R, const double cosElt, const double sinElt, const uint axis, bool startFromZero) {
    if (startFromZero) {
        R = Eigen::MatrixXd::Zero(3,3);
    }
    else {
        R = Eigen::MatrixXd::Identity(3,3);
    }

    const double negSinElt = -1.0 * sinElt;

    switch (axis) {
        case 0: // fixed joints
            // NOTE:
            // This is just a simplified implementation!!!
            // The rotation matrix is just identity matrix for all fixed joints of Fetch
            // don't do anything
            return;
        case 1: // rx
            R(1,1) = cosElt;
            R(1,2) = negSinElt;
            R(2,1) = sinElt;
            R(2,2) = cosElt;
            break;
        case 2: // ry
            R(0,0) = cosElt;
            R(0,2) = sinElt;
            R(2,0) = negSinElt;
            R(2,2) = cosElt;
            break;
        case 3: // rz
            R(0,0) = cosElt;
            R(0,1) = negSinElt;
            R(1,0) = sinElt;
            R(1,1) = cosElt;
            break;
        default:
            WARNING_PRINT("Undefined axis");
            throw -1;
    }
}

bool PZsparse::internalCheck() const {
    if (center.rows() != NRows) {
        WARNING_PRINT("PZsparse error: center matrix number of rows not consistent!");
        return false;
    }
    if (center.cols() != NCols) {
        WARNING_PRINT("PZsparse error: center matrix number of columns not consistent!");
        return false;
    }
    if (independent.rows() != NRows) {
        WARNING_PRINT("PZsparse error: independent generator matrix number of rows not consistent!");
        return false;
    }
    if (independent.cols() != NCols) {
        WARNING_PRINT("PZsparse error: independent generator matrix number of columns not consistent!");
        return false;
    }
    for (uint i = 0; i < independent.rows(); i++) {
        for (uint j = 0; j < independent.cols(); j++) {
            if (independent(i, j) < 0) {
                WARNING_PRINT("PZsparse error: independent generator matrix has negative entry!");
                return false;
            }
        }
    }
    return true;
}

void PZsparse::simplify() {
    assert(internalCheck());

    sort(polynomial.begin(), polynomial.end(), Monomial_sorter_degree);

    Eigen::MatrixXd reduce_amount(NRows, NCols); 
    reduce_amount.setZero();

    vector<Monomial> polynomial_new;
    polynomial_new.reserve(polynomial.size());

    uint i = 0;
    while (i < polynomial.size()) {
        uint j;
        uint64_t degree = polynomial[i].degree;

        for (j = i + 1; j < polynomial.size(); j++) {
            if (polynomial[j].degree != degree) {
                break;
            }

            polynomial[i].coeff += polynomial[j].coeff;
        }

        Eigen::MatrixXd temp = polynomial[i].coeff;
        if (temp.norm() <= SIMPLIFY_THRESHOLD) {
            reduce_amount += temp.cwiseAbs();
        }
        else {
            polynomial_new.emplace_back(polynomial[i]);
        }

        i = j;
    }

    // for (uint i = 0; i < polynomial.size(); i++) {
    //     if (polynomial[i].coeff.norm() == 0) {
    //         continue;
    //     }

    //     uint64_t degree = polynomial[i].degree;

    //     for (uint j = 0; j < polynomial.size(); j++) {
    //         if (j == i || polynomial[j].coeff.norm() == 0) {
    //             continue;
    //         }
    //         if (polynomial[j].degree == degree) {
    //             polynomial[i].coeff += polynomial[j].coeff;
    //             polynomial[j].coeff.setZero();
    //         }
    //     }

    //     Eigen::MatrixXd temp = polynomial[i].coeff;
    //     if (temp.norm() <= SIMPLIFY_THRESHOLD) {
    //         reduce_amount += temp.cwiseAbs();
    //     }
    //     else {
    //         polynomial_new.emplace_back(polynomial[i]);
    //     }
    // }

    polynomial = polynomial_new;

    if (reduce_amount.norm() != 0) {
        independent = independent + reduce_amount;
    }
}

void PZsparse::reduce() {
    assert(internalCheck());

    vector<Monomial> polynomial_new;
    polynomial_new.reserve(polynomial.size());

    for (auto it : polynomial) {
        if (it.degree < max_hash_dependent_k_only) { // only dependent on k
            polynomial_new.emplace_back(it.coeff, it.degree);
        }
        else {
            independent += it.coeff.cwiseAbs();
        }
    }

    polynomial = polynomial_new;
}

Eigen::MatrixXd PZsparse::reduce_link_PZ() {
    assert(internalCheck());
    assert(NRows == 3 && NCols == 1);

    Eigen::MatrixXd link_independent_generators(3, 3 + 3);
    link_independent_generators.setZero();

    vector<Monomial> polynomial_new;
    polynomial_new.reserve(polynomial.size());

    int j = 0;

    for (auto it : polynomial) {
        if (it.degree < max_hash_dependent_k_only) { // only dependent on k
            polynomial_new.emplace_back(it.coeff, it.degree);
        }
        else if (it.degree < max_hash_dependent_k_links_only && (it.degree & dependent_k_mask) == 0) { // only dependent on link x, y, z generators
            assert(j < 3);
            link_independent_generators.col(j++) = it.coeff;
        }
        else {
            independent += it.coeff.cwiseAbs();
        }
    }

    polynomial = polynomial_new;

    link_independent_generators(0, 3) = independent(0);
    link_independent_generators(1, 4) = independent(1);
    link_independent_generators(2, 5) = independent(2);

    return link_independent_generators;
}

MatrixXInt PZsparse::slice(const double* factor) {
    assert(internalCheck());
    MatrixXInt res(NRows, NCols);
    Eigen::MatrixXd res_center = center;
    Eigen::MatrixXd res_radius = independent;

    for (auto it : polynomial) {
        Eigen::MatrixXd resTemp = it.coeff;

        if (it.degree < (1 << (2 * NUM_FACTORS))) { // only dependent on k
            convertHashToDegree(it.degree);

            for (uint j = 0; j < NUM_FACTORS; j++) {
                resTemp *= pow(factor[j], degreeArray[j]);
            }

            res_center += resTemp;
        }
        else { // this line should never be triggered if you run reduce first
            res_radius += resTemp.cwiseAbs();
        }
    }

    for (uint i = 0; i < NRows; i++) {
        for (uint j = 0; j < NCols; j++) {
            res(i,j) = Interval(res_center(i,j) - res_radius(i,j),
                                res_center(i,j) + res_radius(i,j));
        }
    }

    return res;
}

void PZsparse::slice(Eigen::MatrixXd* gradient, const double* factor) {
    assert(internalCheck());

    for (uint k = 0; k < NUM_FACTORS; k++) {
        gradient[k] = Eigen::MatrixXd::Zero(NRows, NCols);
    }

    Eigen::Array<Eigen::MatrixXd, NUM_FACTORS, 1> resTemp;

    for (auto it : polynomial) {
        if (it.degree <= (1 << (2 * NUM_FACTORS))) { // only dependent on k
            for (uint k = 0; k < NUM_FACTORS; k++) {
                resTemp[k] = it.coeff;
            }

            convertHashToDegree(it.degree);

            for (uint j = 0; j < NUM_FACTORS; j++) {
                for (uint k = 0; k < NUM_FACTORS; k++) {
                    if (j == k) { // differentiate this!
                        if (degreeArray[j] == 0) { // monomial unrelated to k
                            resTemp[k] = Eigen::MatrixXd::Zero(NRows, NCols);
                        }
                        else {
                            resTemp[k] *= degreeArray[j] * pow(factor[j], degreeArray[j] - 1);
                        }
                    }
                    else {
                        resTemp[k] *= pow(factor[j], degreeArray[j]);
                    }
                }
            }

            for (uint k = 0; k < NUM_FACTORS; k++) {
                gradient[k] += resTemp[k];
            }
        }
    }
}

void PZsparse::slice(Eigen::Vector3d* gradient, const double* factor) {
    assert(NRows == 3 && NCols == 1);
    assert(internalCheck());

    for (uint k = 0; k < NUM_FACTORS; k++) {
        gradient[k].setZero();
    }

    Eigen::Array<Eigen::Vector3d, NUM_FACTORS, 1> resTemp;

    for (auto it : polynomial) {
        if (it.degree <= (1 << (2 * NUM_FACTORS))) { // only dependent on k
            for (uint k = 0; k < NUM_FACTORS; k++) {
                resTemp[k] = it.coeff;
            }

            convertHashToDegree(it.degree);

            for (uint j = 0; j < NUM_FACTORS; j++) {
                for (uint k = 0; k < NUM_FACTORS; k++) {
                    if (j == k) { // differentiate this!
                        if (degreeArray[j] == 0) { // monomial unrelated to k
                            resTemp[k].setZero();
                        }
                        else {
                            resTemp[k] *= degreeArray[j] * pow(factor[j], degreeArray[j] - 1);
                        }
                    }
                    else {
                        resTemp[k] *= pow(factor[j], degreeArray[j]);
                    }
                }
            }

            for (uint k = 0; k < NUM_FACTORS; k++) {
                gradient[k] += resTemp[k];
            }
        }
    }
}

void PZsparse::slice(double* gradient, const double* factor) {
    assert(internalCheck());
    assert(NRows == 1 && NCols == 1);

    memset(gradient, 0, NUM_FACTORS * sizeof(double));

    double resTemp[NUM_FACTORS] = {0};

    for (auto it : polynomial) {
        if (it.degree <= (1 << (2 * NUM_FACTORS))) { // only dependent on k
            for (uint k = 0; k < NUM_FACTORS; k++) {
                resTemp[k] = it.coeff(0);
            }

            convertHashToDegree(it.degree);

            for (uint j = 0; j < NUM_FACTORS; j++) {
                for (uint k = 0; k < NUM_FACTORS; k++) {
                    if (j == k) { // differentiate this!
                        if (degreeArray[j] == 0) { // monomial unrelated to k
                            resTemp[k] = 0;
                        }
                        else {
                            resTemp[k] *= degreeArray[j] * pow(factor[j], degreeArray[j] - 1);
                        }
                    }
                    else {
                        resTemp[k] *= pow(factor[j], degreeArray[j]);
                    }
                }
            }

            for (uint k = 0; k < NUM_FACTORS; k++) {
                gradient[k] += resTemp[k];
            }
        }
    }
}

MatrixXInt PZsparse::toInterval() {
    assert(internalCheck());

    MatrixXInt res(NRows, NCols);
    Eigen::MatrixXd res_center = center;
    Eigen::MatrixXd res_radius = independent;

    for (auto it : polynomial) {
        res_radius += it.coeff.cwiseAbs();
    }

    for (uint i = 0; i < NRows; i++) {
        for (uint j = 0; j < NCols; j++) {
            res(i,j) = Interval(res_center(i,j) - res_radius(i,j),
                                res_center(i,j) + res_radius(i,j));
        }
    }

    return res;
}

void PZsparse::convertHashToDegree(uint64_t degree) {
    for (uint64_t i = 0; i < NUM_FACTORS * 6; i++) {
        degreeArray[i] = degree & DEGREE_MASK[i];    
        degree >>= MOVE_BIT_INC[i];
    }

    return;
}

uint64_t convertDegreeToHash(const uint64_t* degreeArray) {
    uint64_t degree = 0;
    uint64_t move_bit = 0;

    for (uint64_t i = 0; i < NUM_FACTORS * 6; i++) {
        if (degreeArray[i] > 1) {
            WARNING_PRINT("degree can not be larger than 1!");
            throw;
        }

        degree += (degreeArray[i] << move_bit);

        move_bit += MOVE_BIT_INC[i];
    }

    return degree;
}

std::ostream& operator<<(std::ostream& os, PZsparse& a) {
    // if (independent_only) {
    //     Interval temp = center + independent;
    //     cout << "[ " << temp.lower() << ", " << temp.upper() << " ]\n\n";
    //     return;
    // }

    os << a.center << " +...\n";

    for (auto it : a.polynomial) {
        os << '(' << it.coeff << ')';
        
        a.convertHashToDegree(it.degree);
        
        os << " * k^(";
        for (uint j = 0; j < NUM_FACTORS; j++) {
            os << a.degreeArray[j];
        }
        os << ") ";

        os << " * qde^(";
        for (uint j = 0; j < NUM_FACTORS; j++) {
            os << a.degreeArray[j + NUM_FACTORS * 1];
        }
        os << ") ";

        os << " * qdae^(";
        for (uint j = 0; j < NUM_FACTORS; j++) {
            os << a.degreeArray[j + NUM_FACTORS * 2];
        }
        os << ") ";

        os << " * qddae^(";
        for (uint j = 0; j < NUM_FACTORS; j++) {
            os << a.degreeArray[j + NUM_FACTORS * 3];
        }
        os << ") ";

        os << " * cosqe^(";
        for (uint j = 0; j < NUM_FACTORS; j++) {
            os << a.degreeArray[j + NUM_FACTORS * 4];
        }
        os << ") ";

        os << " * sinqe^(";
        for (uint j = 0; j < NUM_FACTORS; j++) {
            os << a.degreeArray[j + NUM_FACTORS * 5];
        }
        os << ") ";

        os << " +...\n";
    }

    os << "[ " << -a.independent << ", " << a.independent << " ]\n\n";

    return os;
}

std::ostream& operator<<(std::ostream& os, const MatrixXInt& a) {
    for (uint i = 0; i < a.rows(); i++) {
        for (uint j = 0; j < a.cols(); j++) {
            os << "[ " << a(i,j).lower() << ", " << a(i,j).upper() << "] ";
        }
        os << '\n';
    }

    return os;
}

/*
Arithmetic
*/

PZsparse PZsparse::operator() (int row_id, int col_id) const {
    assert(internalCheck());
    assert(row_id < NRows);
    assert(col_id < NCols);

    PZsparse res(1, 1);
    
    res.center = center.block(row_id, col_id, 1, 1);

    res.polynomial.reserve(polynomial.size());

    for (auto it : polynomial) {
        res.polynomial.emplace_back(it.coeff.block(row_id, col_id, 1, 1), it.degree);
    }

    res.independent = independent.block(row_id, col_id, 1, 1);

    return res;

}

PZsparse PZsparse::operator=(const double a) {
    NRows = 1;
    NCols = 1;
    center.resize(NRows, NCols);
    center(0) = a;
    polynomial.clear();
    independent = Eigen::MatrixXd::Zero(NRows, NCols);
    return *this;
}

// PZsparse PZsparse::operator=(const Interval& a) {
//     center = getCenter(a);
//     polynomial.clear();
//     independent = a - center;
//     return *this;
// }

PZsparse PZsparse::operator=(const PZsparse& a) {
    NRows = a.NRows;
    NCols = a.NCols;
    center = a.center;
    polynomial = a.polynomial;
    independent = a.independent;
    return *this;
}

PZsparse PZsparse::operator-() {
    assert(internalCheck());

    PZsparse res(NRows, NCols);
    
    res.center = -center;

    res.polynomial.reserve(polynomial.size());

    for (auto it : polynomial) {
        res.polynomial.emplace_back(-it.coeff, it.degree);
    }

    res.independent = -independent;

    return res;
}

PZsparse PZsparse::operator+(const PZsparse& a) {
    assert(internalCheck());
    assert(a.NRows == NRows || a.NCols == NCols); // check if they are add-able

    PZsparse res(NRows, NCols);

    res.center = center + a.center;

    res.polynomial.reserve(polynomial.size() + a.polynomial.size());

    res.polynomial.insert(res.polynomial.end(), polynomial.begin(), polynomial.end());

    for (auto it : a.polynomial) {
        res.polynomial.push_back(it);
    }

    res.independent = independent + a.independent;

    res.simplify();
    
    return res;
}

PZsparse PZsparse::operator+(const double a) {
    assert(internalCheck());

    PZsparse res = *this;

    for (uint i = 0; i < NRows; i++) {
        for (uint j = 0; j < NCols; j++) {
            res.center(i,j) += a;
        }
    }

    return res;
}

PZsparse operator+(const double a, const PZsparse& b) {
    assert(b.internalCheck());

    PZsparse res = b;

    for (uint i = 0; i < b.NRows; i++) {
        for (uint j = 0; j < b.NCols; j++) {
            res.center(i,j) += a;
        }
    }

    return res;
}

PZsparse PZsparse::operator+=(const PZsparse& a) {
    assert(internalCheck());
    assert(a.NRows == NRows || a.NCols == NCols); // check if they are add-able

    center += a.center;

    polynomial.reserve(polynomial.size() + a.polynomial.size());

    for (auto it : a.polynomial) {
        polynomial.push_back(it);
    }

    independent += a.independent;

    simplify();
    
    return *this;
}

PZsparse PZsparse::operator-(const PZsparse& a) {
    assert(internalCheck());    
    assert(a.NRows == NRows || a.NCols == NCols); // check if they are add-able

    PZsparse res(NRows, NCols);

    res.center = center - a.center;

    res.polynomial.reserve(polynomial.size() + a.polynomial.size());

    res.polynomial.insert(res.polynomial.end(), polynomial.begin(), polynomial.end());

    for (auto it : a.polynomial) {
        res.polynomial.emplace_back(-it.coeff, it.degree);
    }

    res.independent = independent + a.independent;

    res.simplify();
    
    return res;
}

PZsparse PZsparse::operator-(const double a) {
    assert(internalCheck());

    PZsparse res = *this;

    for (uint i = 0; i < NRows; i++) {
        for (uint j = 0; j < NCols; j++) {
            res.center(i,j) -= a;
        }
    }

    return res;
}

PZsparse operator-(const double a, const PZsparse& b) {
    assert(b.internalCheck());

    PZsparse res = b;

    for (uint i = 0; i < b.NRows; i++) {
        for (uint j = 0; j < b.NCols; j++) {
            res.center(i,j) -= a;
        }
    }

    return res;
}

PZsparse PZsparse::operator*(const PZsparse& a) {
    assert(internalCheck());
    assert(NCols == a.NRows || (NRows == 1 && NCols == 1) || (a.NRows == 1 && a.NCols == 1));

    PZsparse res;

    if (NRows == 1 && NCols == 1) {
        res.NRows = a.NRows;
        res.NCols = a.NCols;
    }
    else if (a.NRows == 1 && a.NCols == 1) {
        res.NRows = NRows;
        res.NCols = NCols;
    }
    else {
        res.NRows = NRows;
        res.NCols = a.NCols;
    }

    // center * center
    if (NRows == 1 && NCols == 1) {
        res.center = center(0) * a.center;
    }
    else if (a.NRows == 1 && a.NCols == 1) {
        res.center = center * a.center(0);
    }
    else {
        res.center = center * a.center;
    }

    res.polynomial.reserve(polynomial.size() + a.polynomial.size() + polynomial.size() * a.polynomial.size());
    // a.center * polynomial
    for (auto it : polynomial) {
        if (NRows == 1 && NCols == 1) {
            res.polynomial.emplace_back(it.coeff(0) * a.center, it.degree);
        }
        else if (a.NRows == 1 && a.NCols == 1) {
            res.polynomial.emplace_back(it.coeff * a.center(0), it.degree);
        }
        else {
            res.polynomial.emplace_back(it.coeff * a.center, it.degree);
        }
    }

    // center * a.polynomial
    for (auto it : a.polynomial) {
        if (NRows == 1 && NCols == 1) {
            res.polynomial.emplace_back(center(0) * it.coeff, it.degree);
        }
        else if (a.NRows == 1 && a.NCols == 1) {
            res.polynomial.emplace_back(center * it.coeff(0), it.degree);
        }
        else {
            res.polynomial.emplace_back(center * it.coeff, it.degree);
        }
    }

    // polynomial * a.polynomial (degree for each factor shouldn't be larger than 1)
    // Eigen::MatrixXd reduce_amount_1 = Eigen::MatrixXd::Zero(NRows, a.NCols);

    for (auto it1 : polynomial) {
        for (auto it2 : a.polynomial) {
            Eigen::MatrixXd multiply_coeff = it1.coeff * it2.coeff;

            if (NRows == 1 && NCols == 1) {
                multiply_coeff = it1.coeff(0) * it2.coeff;
            }
            else if (a.NRows == 1 && a.NCols == 1) {
                multiply_coeff = it1.coeff * it2.coeff(0);
            }
            else {
                multiply_coeff = it1.coeff * it2.coeff;
            }

            // Do not have to check carry
            // if we already know the maximum degree in the polynomial
            res.polynomial.emplace_back(multiply_coeff, it1.degree + it2.degree);
        }
    }

    // a.independent * (center + polynomial)
    Eigen::MatrixXd reduce_amount_2 = center.cwiseAbs();

    for (auto it : polynomial) {
        reduce_amount_2 += it.coeff.cwiseAbs();
    }

    if (NRows == 1 && NCols == 1) {
        reduce_amount_2 = reduce_amount_2(0) * a.independent;
    }
    else if (a.NRows == 1 && a.NCols == 1) {
        reduce_amount_2 *= a.independent(0);
    }
    else {
        reduce_amount_2 *= a.independent;
    }
    
    // independent * (a.center + a.polynomial)
    Eigen::MatrixXd reduce_amount_3 = a.center.cwiseAbs();

    for (auto it : a.polynomial) {
        reduce_amount_3 += it.coeff.cwiseAbs();
    }
    
    if (NRows == 1 && NCols == 1) {
        reduce_amount_3 = independent(0) * reduce_amount_3;
    }
    else if (a.NRows == 1 && a.NCols == 1) {
        reduce_amount_3 = independent * reduce_amount_3(0);
    }
    else {
        reduce_amount_3 = independent * reduce_amount_3;
    }

    // independent * a.independent + add reduced intervals
    Eigen::MatrixXd reduce_amount = reduce_amount_2 + reduce_amount_3;

    if (NRows == 1 && NCols == 1) {
        res.independent = independent(0) * a.independent + reduce_amount;
    }
    else if (a.NRows == 1 && a.NCols == 1) {
        res.independent = independent * a.independent(0) + reduce_amount;
    }
    else {
        res.independent = independent * a.independent + reduce_amount;
    }

    res.simplify();
    
    return res;
}

PZsparse PZsparse::operator*(const double a) {
    assert(internalCheck());

    PZsparse res(NRows, NCols);

    res.center = center * a;

    res.polynomial.reserve(polynomial.size());

    for (auto it : polynomial) {
        res.polynomial.emplace_back(a * it.coeff, it.degree);
    }

    res.independent = independent * fabs(a);

    return res;
}

PZsparse operator*(const double a, const PZsparse& b) {
    assert(b.internalCheck());

    PZsparse res(b.NRows, b.NCols);

    res.center = b.center * a;

    res.polynomial.reserve(b.polynomial.size());

    for (auto it : b.polynomial) {
        res.polynomial.emplace_back(a * it.coeff, it.degree);
    }

    res.independent = b.independent * fabs(a);

    return res;
}

PZsparse PZsparse::operator/(const double a) {
    assert(internalCheck());

    PZsparse res(NRows, NCols);

    res.center = center / a;

    res.polynomial.reserve(polynomial.size());

    for (auto it : polynomial) {
        res.polynomial.emplace_back(it.coeff / a, it.degree);
    }

    res.independent = independent / fabs(a);

    return res;
}

PZsparse PZsparse::transpose() {
    assert(internalCheck());

    PZsparse res(NCols, NRows);

    res.center = center.transpose();

    res.polynomial.reserve(polynomial.size());

    for (auto it : polynomial) {
        res.polynomial.emplace_back(it.coeff.transpose(), it.degree);
    }

    res.independent = independent.transpose();

    return res;
}

void PZsparse::addOneDimPZ(const PZsparse& a, uint row_id, uint col_id) {
    assert(internalCheck());
    assert(a.NRows == 1 && a.NCols == 1);
    assert(row_id < NRows);
    assert(col_id < NCols);

    center(row_id, col_id) += a.center(0);

    for (auto it : a.polynomial) {
        Eigen::MatrixXd temp_coeff = Eigen::MatrixXd::Zero(NRows, NCols);
        temp_coeff(row_id, col_id) = it.coeff(0);
        polynomial.emplace_back(temp_coeff, it.degree);
    }

    independent(row_id, col_id) += a.independent(0);

    simplify();
}

PZsparse stack(const PZsparseArray& a) {
    assert(a.cols() == 1);

    for (uint i = 0; i < a.rows(); i++) {
        assert(a(i, 0).NRows == 1 && a(i, 0).NCols == 1);
    }

    PZsparse res(a.rows(), 1);

    for (uint i = 0; i < a.rows(); i++) {
        res.center(i, 0) = a(i, 0).center(0);
    }

    res.polynomial.reserve(a.rows() * a(0, 0).polynomial.size());
    for (uint i = 0; i < a.rows(); i++) {
        for (auto it : a(i, 0).polynomial) {
            Eigen::MatrixXd temp_coeff = Eigen::MatrixXd::Zero(a.rows(), 1);
            temp_coeff(i) = it.coeff(0);
            res.polynomial.emplace_back(temp_coeff, it.degree);
        }
    }

    for (uint i = 0; i < a.rows(); i++) {
        res.independent(i, 0) = a(i, 0).independent(0);
    }

    res.simplify();

    return res;
}

PZsparse cross(const Eigen::MatrixXd& a, const PZsparse& b) {
    assert(a.rows() == 3 && a.cols() == 1 && b.NRows == 3 && b.NCols == 1);

    PZsparseArray res(3, 1);

    PZsparse b0 = b(0, 0);
    PZsparse b1 = b(1, 0);
    PZsparse b2 = b(2, 0);

    res(0, 0) = a(1, 0) * b2 - a(2, 0) * b1;
    res(1, 0) = a(2, 0) * b0 - a(0, 0) * b2;
    res(2, 0) = a(0, 0) * b1 - a(1, 0) * b0;

    return stack(res);
}

PZsparse cross(const PZsparse& a, const PZsparse& b) {
    assert(a.NRows == 3 && a.NCols == 1 && b.NRows == 3 && b.NCols == 1);

    PZsparseArray res(3, 1);

    PZsparse a0 = a(0, 0);
    PZsparse a1 = a(1, 0);
    PZsparse a2 = a(2, 0);
    PZsparse b0 = b(0, 0);
    PZsparse b1 = b(1, 0);
    PZsparse b2 = b(2, 0);

    res(0, 0) = a1 * b2 - a2 * b1;
    res(1, 0) = a2 * b0 - a0 * b2;
    res(2, 0) = a0 * b1 - a1 * b0;

    return stack(res);
}

PZsparse cross(const PZsparse& a, const Eigen::MatrixXd& b) {
    assert(a.NRows == 3 && a.NCols == 1 && b.rows() == 3 && b.cols() == 1);

    PZsparseArray res(3, 1);

    PZsparse a0 = a(0, 0);
    PZsparse a1 = a(1, 0);
    PZsparse a2 = a(2, 0);

    res(0, 0) = a1 * b(2, 0) - a2 * b(1, 0);
    res(1, 0) = a2 * b(0, 0) - a0 * b(2, 0);
    res(2, 0) = a0 * b(1, 0) - a1 * b(0, 0);

    return stack(res);
}

#endif